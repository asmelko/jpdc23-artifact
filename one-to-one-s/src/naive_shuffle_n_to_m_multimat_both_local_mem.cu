#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "row_distribution.cuh"
#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross::local_mem {

namespace {

constexpr dsize_t left_matrices_per_thread_limit = SHUFFLE_N_TO_M_MULTIMAT_BOTH_LOCAL_MEM_LEFT_MATRICES_PER_THREAD_LIMIT;
constexpr dsize_t right_matrices_per_thread_limit = SHUFFLE_N_TO_M_MULTIMAT_BOTH_LOCAL_MEM_RIGHT_MATRICES_PER_THREAD_LIMIT;

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;
    dsize_t num_right_matrices;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t num_right_matrices
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), output_pos(output_pos),
        matrix_size(matrix_size), search_size(search_size), num_right_matrices(num_right_matrices) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size,
        num_right_matrices
    );
}

template<dsize_t NUM_LEFTS, dsize_t NUM_RIGHTS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl(
    const cg::thread_block_tile <WARP_SIZE>& warp,
    const warp_shuffle_impl_args<T, RES>& args
) {
    // Compute the given shift for num_rights right matrices
    RES sum[NUM_LEFTS * NUM_RIGHTS];
    #pragma unroll
    for (dsize_t i = 0; i < NUM_LEFTS * NUM_RIGHTS; ++i) {
        sum[i] = 0;
    }

    for (dsize_t warp_y_right = args.warp_right_start.y; warp_y_right < args.warp_right_end.y; warp_y_right += 1) {
        // In y axis, both max and min shift are equal in the current implementation
        dsize_t warp_y_left = warp_y_right + args.warp_min_shift.y;

        const dsize_t right_row_offset = warp_y_right * args.matrix_size.x;
        const T* left_row = args.left + warp_y_left * args.matrix_size.x;
        const T* right_row = args.right + right_row_offset;

        int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

        // Preload the first values from left matrix
        T thread_left_bottom[NUM_LEFTS];
        #pragma unroll
        for (dsize_t l = 0; l < NUM_LEFTS; ++l) {
            thread_left_bottom[l] = load_with_bounds_check(
                left_row + l * args.matrix_size.area(),
                warp_x_left + warp.thread_rank(),
                args.matrix_size.x
            );
        }


        for (
            dsize_t warp_x_right = args.warp_right_start.x;
            warp_x_right < args.warp_right_end.x;
            warp_x_right += warp.size(), warp_x_left += warp.size()
            ) {

            // Load next warp_size values
            // Load 0 if out of bounds

            // Right index will always be greater than 0 as we only
            // iterate over part of the matrix
            dsize_t right_idx = warp_x_right + warp.thread_rank();

            // Left index might be out of bounds even below 0, depending on the shift
            // It is also reading warp.size() next values, as we have warp.size() values already loaded
            // from the initialization before the for loop
            int left_idx = warp_x_left + warp.thread_rank() + warp.size();

            // Load values from num_rights right matrices
            T thread_right[NUM_RIGHTS];
            #pragma unroll
            for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
                // TODO: Either do bounds check or limit the for loop below
                thread_right[r] = load_with_bounds_check(
                    right_row + r * args.matrix_size.area(), right_idx, args.matrix_size.x
                );
            }

            T thread_left_top[NUM_LEFTS];
            #pragma unroll
            for (dsize_t l = 0; l < NUM_LEFTS; ++l) {
                thread_left_top[l] = load_with_bounds_check(
                    left_row + l * args.matrix_size.area(),
                    left_idx,
                    args.matrix_size.x
                );
            }

            // TODO: Maybe pragma unroll?
            for (dsize_t i = 0; i < warp.size(); ++i) {
                // TODO: Merge into a single for loop which may be easier for compiler to unroll
                //  and derive the r and l variables using modulo and division
                #pragma unroll
                for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
                    // Broadcast
                    auto right_val = warp.shfl(thread_right[r], i);

                    #pragma unroll
                    for (dsize_t l = 0; l < NUM_LEFTS; ++l) {
                        // No need to mask, if either values is out of bounds the value will be 0
                        sum[l * NUM_RIGHTS + r] += thread_left_bottom[l] * right_val;
                    }
                }

                #pragma unroll
                for (dsize_t l = 0; l < NUM_LEFTS; ++l) {
                    // Shuffle does modulo srcLane automatically
                    // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                    //  making it behave as one continuous buffer
                    thread_left_bottom[l] = warp.shfl(
                        warp.thread_rank() != 0 ? thread_left_bottom[l] : thread_left_top[l],
                        warp.thread_rank() + 1
                    );
                    thread_left_top[l] = warp.shfl_down(thread_left_top[l], 1);
                }
            }
        }
    }

    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        auto output_offset = args.output_pos.linear_idx(args.search_size.x);
        // TODO: Merge into a single for loop which may be easier for compiler to unroll
        //  and derive the r and l variables using modulo and division
        #pragma unroll
        for (dsize_t l = 0; l < NUM_LEFTS; ++l) {
            #pragma unroll
            for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
                T* matrix = args.out + (l * args.num_right_matrices + r) * args.search_size.area();
                if (ATOMIC) {
                    atomicAdd(matrix + output_offset, sum[l * NUM_RIGHTS + r]);
                } else {
                    matrix[output_offset] = sum[l * NUM_RIGHTS + r];
                }
            }
        }
    }
}

// TODO: Is this correct?
template<dsize_t NUM_LEFTS, dsize_t NUM_RIGHTS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl_dispatch_num_rights(
    const cg::thread_block_tile <WARP_SIZE>& warp,
    dsize_t thread_num_rights,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_RIGHTS == 0) {
        // Silence the unused parameter warning
        (void)warp;
        (void)thread_num_rights;
        (void)args;
        assert(false);
    } else {
        if (NUM_RIGHTS == thread_num_rights) {
            warp_shuffle_impl<NUM_LEFTS, NUM_RIGHTS, ATOMIC>(
                warp,
                args
            );
        } else {
            warp_shuffle_impl_dispatch_num_rights<NUM_LEFTS, NUM_RIGHTS - 1, ATOMIC>(
                warp,
                thread_num_rights,
                args
            );
        }
    }
}

// TODO: Is this correct?
template<dsize_t NUM_LEFTS, dsize_t NUM_RIGHTS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl_dispatch_num_lefts(
    const cg::thread_block_tile <WARP_SIZE>& warp,
    dsize_t thread_num_lefts,
    dsize_t thread_num_rights,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_LEFTS == 0) {
        // Silence the unused parameter warning
        (void)warp;
        (void)thread_num_lefts;
        (void)thread_num_rights;
        (void)args;
        assert(false);
    } else {
        if (NUM_LEFTS == thread_num_lefts) {
            warp_shuffle_impl_dispatch_num_rights<NUM_LEFTS, NUM_RIGHTS, ATOMIC>(
                warp,
                thread_num_rights,
                args
            );
        } else {
            warp_shuffle_impl_dispatch_num_lefts<NUM_LEFTS - 1, NUM_RIGHTS, ATOMIC>(
                warp,
                thread_num_lefts,
                thread_num_rights,
                args
            );
        }
    }
}

/**
 * For description of the functionality implemented by this kernel, see ccn_warp_shuffle kernel.
 * This kernel adds distribution of rows of a single shift between multiple threads.
 *
 * @tparam T
 * @tparam RES
 * @param left
 * @param right
 * @param out
 * @param matrix_size
 * @param search_size
 */
template<dsize_t MAX_LEFT_MATRICES_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename DIST, typename T, typename RES>
__global__ void ccn_shuffle_n_to_m_multimat_both_work_distribution(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t max_rows_per_thread
) {

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    // Matrix group is the group of right matrices (of at most right_matrices_per_thread matrices)
    // for which current thread computes the given shift
    // All warps in a block process the same 32 shifts in the x axis, but on different rows
    // so warps in the first block compute shifts 0-31, warps in the second block compute shifts 32-63 etc.
    // So each matrix_group needs to have as many threads as there are shifts in the x axis
    // so number of shifts in the x axis / warp_size
    // TODO: This is precomputed on CPU so we could pass it from there
    dsize_t blocks_per_matrix_group = div_up(search_size.x, warp_size);

    // Which matrix group this block and all its warps will compute
    // THe X axis of block index encodes the shift in x axis together with the left matrix group the thread belongs to
    dsize_t left_matrix_group_idx = ctb.group_index().x / blocks_per_matrix_group;
    dsize_t right_matrix_group_idx = ctb.group_index().y;

    // Offset of the current block and all of its warps in its matrix group
    // This corresponds to the position to write to in the output and the shift
    // to compute
    dsize_t matrix_group_block_offset = ctb.group_index().x % blocks_per_matrix_group;
    dsize_t warp_output_x_offset = matrix_group_block_offset * warp_size;

    // Index of the first matrix in the group processed by the current thread
    dsize_t left_matrix_group_start_idx = left_matrix_group_idx * MAX_LEFT_MATRICES_PER_THREAD;
    dsize_t right_matrix_group_start_idx = right_matrix_group_idx * MAX_RIGHT_MATRICES_PER_THREAD;

    // Distribute rows of a single shift between multiple workers,
    // in this case threads
    // Return the assigned output row (which corresponds to a shift),
    // together with the number of workers computing this shift and
    // index of the current worker in range [0, number_of_workers_for_shift)
    assigned_work work = DIST::distribute_rows(
        ctb.group_index().z * ctb.group_dim().y + ctb.thread_index().y,
        max_rows_per_thread,
        matrix_size.y,
        search_size.y
    );

    // All threads of a warp should share the same worker_idx and workers_for_row
    // so either the whole warp continues or exists
    if (work.worker_idx >= work.workers_for_row) {
        return;
    }

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos = dsize2_t{
        warp_output_x_offset,
        work.output_row
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x)) - static_cast<int>(half_search_size.x),
        static_cast<int>(min(last_warp_thread_out_pos.y, search_size.y)) - static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    // All threads in a warp process the same range of rows, so warp_min_shift.y and warp_max_shift.y are the same
    // Multiple threads from different warps may compute the same shift
    // These values are shared for all workers computing the same shift
    dsize_t shared_y_right_start = max(-warp_min_shift.y, 0);
    dsize_t shared_y_right_end = min(matrix_size.y - warp_max_shift.y, matrix_size.y);

    dsize_t shared_overlapping_rows = shared_y_right_end - shared_y_right_start;
    dsize_t rows_per_worker = div_up(shared_overlapping_rows, work.workers_for_row);


    // For the current worker
    dsize_t warp_y_right_start = shared_y_right_start + work.worker_idx * rows_per_worker;
    dsize_t warp_y_right_end = min(warp_y_right_start + rows_per_worker, shared_y_right_end);

    dsize_t thread_num_left_matrices = min(num_left_matrices - left_matrix_group_start_idx, MAX_LEFT_MATRICES_PER_THREAD);
    dsize_t thread_num_right_matrices = min(
        num_right_matrices - right_matrix_group_start_idx, MAX_RIGHT_MATRICES_PER_THREAD
    );

    auto args = create_warp_shuffle_impl_args(
        left + left_matrix_group_start_idx * matrix_size.area(),
        right + right_matrix_group_start_idx * matrix_size.area(),
        out + (left_matrix_group_start_idx * num_right_matrices + right_matrix_group_start_idx) * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size,
        num_right_matrices
    );

    warp_shuffle_impl_dispatch_num_lefts<MAX_LEFT_MATRICES_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD, true>(
        warp,
        thread_num_left_matrices,
        thread_num_right_matrices,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args : public kernel_args {
public:
    dsize_t max_left_matrices_per_thread_;
    dsize_t max_right_matrices_per_thread_;
    distribution dist_;

    ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args(const ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args&) = delete;
    ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args& operator=(ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_left_matrices_per_thread,
        dsize_t max_right_matrices_per_thread,
        distribution dist
    ) {
        static ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_left_matrices_per_thread_ = max_left_matrices_per_thread;
        instance.max_right_matrices_per_thread_ = max_right_matrices_per_thread;
        instance.dist_ = dist;
        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_left_matrices_per_thread", std::to_string(max_left_matrices_per_thread_)},
            {"max_right_matrices_per_thread", std::to_string(max_right_matrices_per_thread_)},
            {"work_distribution", to_string(dist_)}
        };
    }

private:
    ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args()
        : kernel_args(),
          max_left_matrices_per_thread_(0),
          max_right_matrices_per_thread_(0),
          dist_(distribution::none)
    { }
};

template<dsize_t MAX_LEFT_MATRICES_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename DIST, typename T, typename RES>
__host__ void ccn_shuffle_n_to_m_multimat_both_work_distribution_right_mat_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
) {
    if constexpr(MAX_RIGHT_MATRICES_PER_THREAD > 0) {
        if (MAX_RIGHT_MATRICES_PER_THREAD == right_matrices_per_thread) {
            dsize_t num_workers = DIST::num_workers(max_rows_per_thread, matrix_size.y, search_size.y);

            // Each row of cuda block corresponds to a single warp for simplified code
            constexpr dsize_t block_x_size = warp_size;

            // There will be total of num_left_matrix_groups * num_right_matrix_groups matrix_groups
            dsize_t num_left_matrix_groups = div_up(num_left_matrices, MAX_LEFT_MATRICES_PER_THREAD);
            dsize_t num_right_matrix_groups = div_up(num_right_matrices, MAX_RIGHT_MATRICES_PER_THREAD);
            // Each shift is still computed by a single thread (in the x axis), so we need as many threads
            // as there are shifts, in each matrix group
            dsize_t blocks_per_matrix_group = div_up(search_size.x, block_x_size);

            dim3 num_threads(block_x_size, warps_per_thread_block);
            dim3 num_blocks(
                // Encodes the shift in x direction and the left matrix group the thread belongs to
                blocks_per_matrix_group * num_left_matrix_groups,
                // Encodes the right matrix group the thread belongs to
                num_right_matrix_groups,
                // Encodes distribution of matrix rows between threads
                div_up(num_workers, num_threads.y)
            );

            ccn_shuffle_n_to_m_multimat_both_work_distribution<MAX_LEFT_MATRICES_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD, DIST><<<num_blocks, num_threads>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_left_matrices,
                num_right_matrices,
                max_rows_per_thread
            );

            ccn_shuffle_n_to_m_multimat_both_work_distribution_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_LEFT_MATRICES_PER_THREAD,
                MAX_RIGHT_MATRICES_PER_THREAD,
                DIST::type
            );
        } else {
            ccn_shuffle_n_to_m_multimat_both_work_distribution_right_mat_dispatch<MAX_LEFT_MATRICES_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD - 1, DIST>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_left_matrices,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                max_rows_per_thread
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_left_matrices;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_matrices_per_thread;
        (void)max_rows_per_thread;
        assert(false);
    }
}

template<dsize_t MAX_LEFT_MATRICES_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename DIST, typename T, typename RES>
__host__ void ccn_shuffle_n_to_m_multimat_both_work_distribution_left_mat_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_matrices_per_thread,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
) {
    if constexpr(MAX_LEFT_MATRICES_PER_THREAD > 0) {
        if (MAX_LEFT_MATRICES_PER_THREAD == left_matrices_per_thread) {
            ccn_shuffle_n_to_m_multimat_both_work_distribution_right_mat_dispatch<MAX_LEFT_MATRICES_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD, DIST>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_left_matrices,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                max_rows_per_thread
            );
        } else {
            ccn_shuffle_n_to_m_multimat_both_work_distribution_left_mat_dispatch<MAX_LEFT_MATRICES_PER_THREAD - 1, MAX_RIGHT_MATRICES_PER_THREAD, DIST>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_left_matrices,
                num_right_matrices,
                warps_per_thread_block,
                left_matrices_per_thread,
                right_matrices_per_thread,
                max_rows_per_thread
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_left_matrices;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)left_matrices_per_thread;
        (void)right_matrices_per_thread;
        (void)max_rows_per_thread;
        assert(false);
    }
}


} // END anonymous namespace

template<typename DIST, typename T, typename RES>
void run_ccn_shuffle_n_to_m_multimat_both_work_distribution(
    const T *__restrict__ left,
    const T *__restrict__ right,
    RES *__restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_matrices_per_thread,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Too many warps per thread block: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (left_matrices_per_thread > left_matrices_per_thread_limit) {
        throw std::runtime_error(
            "Too many left matrices per thread: "s +
            std::to_string(right_matrices_per_thread) +
            " (max "s +
            std::to_string(left_matrices_per_thread_limit) +
            ")"s
        );
    }

    if (right_matrices_per_thread > right_matrices_per_thread_limit) {
        throw std::runtime_error(
            "Too many right matrices per thread: "s +
            std::to_string(right_matrices_per_thread) +
            " (max "s +
            std::to_string(right_matrices_per_thread_limit) +
            ")"s
        );
    }

    ccn_shuffle_n_to_m_multimat_both_work_distribution_left_mat_dispatch<left_matrices_per_thread_limit, right_matrices_per_thread_limit, DIST>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_left_matrices,
        num_right_matrices,
        warps_per_thread_block,
        left_matrices_per_thread,
        right_matrices_per_thread,
        max_rows_per_thread
    );
}

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<triangle_distribution, int, int>(
//     const int *__restrict__ left,
//     const int *__restrict__ right,
//     int *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<triangle_distribution, float, float>(
    const float *__restrict__ left,
    const float *__restrict__ right,
    float *__restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_matrices_per_thread,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
);

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<triangle_distribution, double, double>(
//     const double *__restrict__ left,
//     const double *__restrict__ right,
//     double *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<rectangle_distribution, int, int>(
//     const int *__restrict__ left,
//     const int *__restrict__ right,
//     int *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<rectangle_distribution, float, float>(
    const float *__restrict__ left,
    const float *__restrict__ right,
    float *__restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_matrices_per_thread,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
);

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<rectangle_distribution, double, double>(
//     const double *__restrict__ left,
//     const double *__restrict__ right,
//     double *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<no_distribution, int, int>(
//     const int *__restrict__ left,
//     const int *__restrict__ right,
//     int *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<no_distribution, float, float>(
    const float *__restrict__ left,
    const float *__restrict__ right,
    float *__restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_left_matrices,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_matrices_per_thread,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread
);

// template void run_ccn_shuffle_n_to_m_multimat_both_work_distribution<no_distribution, double, double>(
//     const double *__restrict__ left,
//     const double *__restrict__ right,
//     double *__restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_left_matrices,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t left_matrices_per_thread,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread
// );

}
