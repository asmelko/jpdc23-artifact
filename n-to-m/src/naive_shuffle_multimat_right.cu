#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "row_distribution.cuh"
#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

constexpr dsize_t right_matrices_per_thread_limit = SHUFFLE_MULTIMAT_RIGHT_RIGHT_MATRICES_PER_THREAD_LIMIT;

__device__ void get_matrix_group(
    dsize_t output_size,
    dsize_t ctb_index,
    dsize_t matrices_per_thread,
    dsize_t& warp_output_offset,
    dsize_t& matrix_group_start_idx
) {
    // Matrix group is the group of right matrices (of at most right_matrices_per_thread matrices)
    // for which current thread computes the given shift
    // All warps in a block process the same 32 shifts in the x axis, but on different rows
    // so warps in the first block compute shifts 0-31, warps in the second block compute shifts 32-63 etc.
    // So each matrix_group needs to have as many threads as there are shifts in the x axis
    // so number of shifts in the x axis / warp_size
    // TODO: This is precomputed on CPU so we could pass it from there
    dsize_t blocks_per_matrix_group = div_up(output_size, warp_size);

    // Which matrix group this block and all its warps will compute
    dsize_t matrix_group_idx = ctb_index / blocks_per_matrix_group;
    // Offset of the current block and all of its warps in its matrix group
    // This corresponds to the position to write to in the output and the shift
    // to compute
    dsize_t matrix_group_block_offset = ctb_index % blocks_per_matrix_group;
    warp_output_offset = matrix_group_block_offset * warp_size;

    // Index of the first matrix in the group processed by the current thread
    matrix_group_start_idx = matrix_group_idx * matrices_per_thread;
}

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), output_pos(output_pos),
        matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_RIGHTS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args
) {
    // Compute the given shift for num_rights right matrices
    RES sum[NUM_RIGHTS];
    for (dsize_t i = 0; i < NUM_RIGHTS; ++i) {
        sum[i] = 0;
    }

    for (dsize_t warp_y_right = args.warp_right_start.y; warp_y_right < args.warp_right_end.y; warp_y_right += 1) {
        // In y axis, both max and min shift are equal in the current implementation
        dsize_t warp_y_left = warp_y_right + args.warp_min_shift.y;

        const dsize_t right_row_offset = warp_y_right * args.matrix_size.x;
        const T* left_row = args.left + warp_y_left * args.matrix_size.x;

        int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

        // Preload the first values from left matrix
        T thread_left_bottom = load_with_bounds_check(
            left_row,
            warp_x_left + warp.thread_rank(),
            args.matrix_size.x
        );

        for (
            dsize_t warp_x_right = args.warp_right_start.x;
            warp_x_right < args.warp_right_end.x;
            warp_x_right += warp.size(), warp_x_left += warp.size()
        ) {

            // Load next warp_size values
            // Load 0 if out of bounds

            // Right index will always be greater than 0 as we only
            // iterate over part of the matrix
            dsize_t right_idx = warp_x_right + warp.thread_rank();

            // Left index might be out of bounds even below 0, depending on the shift
            // It is also reading warp.size() next values, as we have warp.size() values already loaded
            // from the initialization before the for loop
            int left_idx = warp_x_left + warp.thread_rank() + warp.size();

            // Load values from num_rights right matrices
            T thread_right[NUM_RIGHTS];
            for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
                const T* matrix_start = args.right + r * args.matrix_size.area();
                const T* row = matrix_start + right_row_offset;
                // TODO: Either do bounds check or limit the for loop below
                thread_right[r] = load_with_bounds_check(row, right_idx, args.matrix_size.x);
            }

            T thread_left_top = load_with_bounds_check(left_row, left_idx, args.matrix_size.x);

            for (dsize_t i = 0; i < warp.size(); ++i) {
                for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
                    // Broadcast
                    auto right_val = warp.shfl(thread_right[r], i);

                    // No need to mask, if either values is out of bounds the value will be 0
                    sum[r] += thread_left_bottom * right_val;
                }

                // Shuffle does modulo srcLane automatically
                // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                //  making it behave as one continuous buffer
                thread_left_bottom = warp.shfl(
                    warp.thread_rank() != 0 ? thread_left_bottom : thread_left_top,
                    warp.thread_rank() + 1
                );
                thread_left_top = warp.shfl_down(thread_left_top, 1);
            }
        }
    }

    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        auto output_offset = args.output_pos.linear_idx(args.search_size.x);
        for (dsize_t r = 0; r < NUM_RIGHTS; ++r) {
            T* matrix = args.out + r * args.search_size.area();
            if (ATOMIC) {
                atomicAdd(matrix + output_offset, sum[r]);
            } else {
                matrix[output_offset] = sum[r];
            }
        }
    }
}

template<dsize_t NUM_RIGHTS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void warp_shuffle_impl_right_mats_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_matrices,
    warp_shuffle_impl_args<T, RES> args
) {
    if constexpr(NUM_RIGHTS == 0) {
        (void)warp;
        (void)num_right_matrices;
        (void)args;
        assert(false);
    } else {
        if (NUM_RIGHTS == num_right_matrices) {
            warp_shuffle_impl<NUM_RIGHTS, ATOMIC>(warp, args);
        } else {
            warp_shuffle_impl_right_mats_dispatch<NUM_RIGHTS - 1, ATOMIC>(
                warp, num_right_matrices, args
            );
        }
    }
}

/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename T, typename RES>
__global__ void ccn_shuffle_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices
) {
    // Initialize by loading a warp worth of data from left matrix
    // as we will be iterating over the left matrix

    // Then broadcast from the right data in sequence from all threads
    // With each broadcast, multiply and sum with the current value from
    // left matrix and then shuffle down the used values from left matrix.
    // Then shuffle the second warp worth of data from left matrix,
    // passing the last thread the value that is shuffled out of the thread 0
    // and would be forgotten
    // basically with warp size 4, it will go
    // 0 1 2 3 0 1 2 3, then 1 2 3 0 1 2 3 x, then 2 3 0 1 2 3 x x,
    // each time broadcasting first from thread 0, then 1, then 2
    // Once we get to 0 1 2 3 x x x x, we load one warp worth of values
    // from both left and right matrices

    // If the shift computed by the current thread does not overlap with the broadcast value
    // that means it tries to read from the left matrix out of bounds and thus will read 0
    // and ignore the broadcast value
    // By shifting the values down, when it reaches the part that overlaps it will receive
    // value shifted from the previous thread

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);


    // Offset in the given output matrix on the x axis
    dsize_t output_x_offset;
    // Index of the first matrix in the group processed by the current thread
    dsize_t matrix_group_start_idx;
    get_matrix_group(
        search_size.x,
        ctb.group_index().x,
        MAX_RIGHT_MATRICES_PER_THREAD,
        output_x_offset,
        matrix_group_start_idx
    );


    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        output_x_offset,
        ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x)) - static_cast<int>(half_search_size.x),
        static_cast<int>(min(last_warp_thread_out_pos.y, search_size.y)) - static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    // All threads in a warp process the same range of rows, so warp_min_shift.y and warp_max_shift.y are the same
    dsize_t warp_y_right_start = max(-warp_min_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_max_shift.y, matrix_size.y);

    dsize_t thread_num_right_matrices = min(num_right_matrices - matrix_group_start_idx, MAX_RIGHT_MATRICES_PER_THREAD);

    auto args = create_warp_shuffle_impl_args(
        left,
        right + matrix_group_start_idx * matrix_size.area(),
        out + matrix_group_start_idx * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size
    );

    warp_shuffle_impl_right_mats_dispatch<MAX_RIGHT_MATRICES_PER_THREAD, false>(
        warp,
        thread_num_right_matrices,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_multimat_right_kernel_args : public kernel_args {
public:
    dsize_t max_right_matrices_per_thread_;

    ccn_shuffle_multimat_right_kernel_args(const ccn_shuffle_multimat_right_kernel_args&) = delete;
    ccn_shuffle_multimat_right_kernel_args& operator=(ccn_shuffle_multimat_right_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_right_matrices_per_thread
    ) {
        static ccn_shuffle_multimat_right_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_right_matrices_per_thread_ = max_right_matrices_per_thread;

        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_right_matrices_per_thread", std::to_string(max_right_matrices_per_thread_)}
        };
    }

private:
    ccn_shuffle_multimat_right_kernel_args()
        : kernel_args(),
          max_right_matrices_per_thread_(0)
    { }
};


template<dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename T, typename RES>
__host__ void ccn_shuffle_multimat_right_right_mats_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread
) {
    if constexpr(MAX_RIGHT_MATRICES_PER_THREAD == 0) {
        // Silence the unused parameter warning
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_matrices_per_thread;
        assert(false);
    } else {
        if (MAX_RIGHT_MATRICES_PER_THREAD == right_matrices_per_thread) {
            dim3 num_threads(warp_size, warps_per_thread_block);

            dsize_t num_matrix_groups = div_up(num_right_matrices, right_matrices_per_thread);
            dsize_t blocks_per_matrix_group = div_up(search_size.x, num_threads.x);


            dim3 num_blocks(
                blocks_per_matrix_group * num_matrix_groups,
                div_up(search_size.y, num_threads.y)
            );

            ccn_shuffle_multimat_right<MAX_RIGHT_MATRICES_PER_THREAD><<<num_blocks, num_threads>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices
            );

            ccn_shuffle_multimat_right_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_RIGHT_MATRICES_PER_THREAD
            );
        } else {
            ccn_shuffle_multimat_right_right_mats_dispatch<MAX_RIGHT_MATRICES_PER_THREAD - 1>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread
            );
        }
    }
}

/**
 * For description of the functionality implemented by this kernel, see ccn_shuffle_multimat_right kernel.
 * This kernel adds distribution of rows of a single shift between multiple threads.
 *
 * @tparam T
 * @tparam RES
 * @param left
 * @param right
 * @param out
 * @param matrix_size
 * @param search_size
 */
template<dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename DIST, typename T, typename RES>
__global__ void ccn_shuffle_multimat_right_work_distribution(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t max_rows_per_thread
) {

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);


    dsize_t warp_output_x_offset;
    // Index of the first matrix in the group processed by the current thread
    dsize_t matrix_group_start_idx;
    get_matrix_group(
        search_size.x,
        ctb.group_index().x,
        MAX_RIGHT_MATRICES_PER_THREAD,
        warp_output_x_offset,
        matrix_group_start_idx
    );

    // Distribute rows of a single shift between multiple workers,
    // in this case threads
    // Return the assigned output row (which corresponds to a shift),
    // together with the number of workers computing this shift and
    // index of the current worker in range [0, number_of_workers_for_shift)
    assigned_work work = DIST::distribute_rows(
        ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y,
        max_rows_per_thread,
        matrix_size.y,
        search_size.y
    );

    // All threads of a warp should share the same worker_idx and workers_for_row
    // so either the whole warp continues or exists
    if (work.worker_idx >= work.workers_for_row) {
        return;
    }

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        warp_output_x_offset,
        work.output_row
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift{
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift{
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x)) - static_cast<int>(half_search_size.x),
        static_cast<int>(min(last_warp_thread_out_pos.y, search_size.y)) - static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    // All threads in a warp process the same range of rows, so warp_min_shift.y and warp_max_shift.y are the same
    // Multiple threads from different warps may compute the same shift
    // These values are shared for all workers computing the same shift
    dsize_t shared_y_right_start = max(-warp_min_shift.y, 0);
    dsize_t shared_y_right_end = min(matrix_size.y - warp_max_shift.y, matrix_size.y);

    dsize_t shared_overlapping_rows = shared_y_right_end - shared_y_right_start;
    dsize_t rows_per_worker = div_up(shared_overlapping_rows, work.workers_for_row);


    // For the current worker
    dsize_t warp_y_right_start = shared_y_right_start + work.worker_idx * rows_per_worker;
    dsize_t warp_y_right_end = min(warp_y_right_start + rows_per_worker, shared_y_right_end);


    dsize_t thread_num_right_matrices = min(num_right_matrices - matrix_group_start_idx, MAX_RIGHT_MATRICES_PER_THREAD);

    auto args = create_warp_shuffle_impl_args(
        left,
        right + matrix_group_start_idx * matrix_size.area(),
        out + matrix_group_start_idx * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        output_pos,
        matrix_size,
        search_size
    );

    warp_shuffle_impl_right_mats_dispatch<MAX_RIGHT_MATRICES_PER_THREAD, true>(
        warp,
        thread_num_right_matrices,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_multimat_right_work_distribution_kernel_args : public kernel_args {
public:
    dsize_t max_right_matrices_per_thread_;
    distribution dist_;

    ccn_shuffle_multimat_right_work_distribution_kernel_args(const ccn_shuffle_multimat_right_work_distribution_kernel_args&) = delete;
    ccn_shuffle_multimat_right_work_distribution_kernel_args& operator=(ccn_shuffle_multimat_right_work_distribution_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_right_matrices_per_thread,
        distribution dist
    ) {
        static ccn_shuffle_multimat_right_work_distribution_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_right_matrices_per_thread_ = max_right_matrices_per_thread;
        instance.dist_ = dist;
        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_right_matrices_per_thread", std::to_string(max_right_matrices_per_thread_)},
            {"work_distribution", to_string(dist_)}
        };
    }

private:
    ccn_shuffle_multimat_right_work_distribution_kernel_args()
        : kernel_args(),
          max_right_matrices_per_thread_(0),
          dist_(distribution::none)
    { }
};

template<dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename DIST, typename T, typename RES>
__host__ void ccn_shuffle_multimat_right_work_distribution_right_mats_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread,
    hipStream_t hip_stream
) {
    if constexpr(MAX_RIGHT_MATRICES_PER_THREAD == 0) {
        // Silence the unused parameter warning
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_matrices_per_thread;
        (void)max_rows_per_thread;
        (void)hip_stream;
        assert(false);
    } else {
        if (MAX_RIGHT_MATRICES_PER_THREAD == right_matrices_per_thread) {
            dsize_t num_workers = DIST::num_workers(max_rows_per_thread, matrix_size.y, search_size.y);

            // Each row of cuda block corresponds to a single warp for simplified code
            constexpr dsize_t block_x_size = warp_size;

            dsize_t num_matrix_groups = div_up(num_right_matrices, right_matrices_per_thread);
            dsize_t blocks_per_matrix_group = div_up(search_size.x, block_x_size);

            dim3 num_threads(block_x_size, warps_per_thread_block);
            dim3 num_blocks(
                blocks_per_matrix_group * num_matrix_groups,
                div_up(num_workers, num_threads.y)
            );

            ccn_shuffle_multimat_right_work_distribution<MAX_RIGHT_MATRICES_PER_THREAD, DIST><<<num_blocks, num_threads, 0, hip_stream>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                max_rows_per_thread
            );

            ccn_shuffle_multimat_right_work_distribution_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_RIGHT_MATRICES_PER_THREAD,
                DIST::type
            );
        } else {
            ccn_shuffle_multimat_right_work_distribution_right_mats_dispatch<MAX_RIGHT_MATRICES_PER_THREAD - 1, DIST>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                max_rows_per_thread,
                hip_stream
            );
        }
    }
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_shuffle_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Too many warps per thread block: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (right_matrices_per_thread == 0 || right_matrices_per_thread > right_matrices_per_thread_limit) {
        throw std::runtime_error(
            "Invalid number of right matrices per thread: "s +
            std::to_string(right_matrices_per_thread) +
            " [1-"s +
            std::to_string(right_matrices_per_thread_limit) +
            "]"s
        );
    }

    ccn_shuffle_multimat_right_right_mats_dispatch<right_matrices_per_thread_limit>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_right_matrices,
        warps_per_thread_block,
        right_matrices_per_thread
    );
}

template<typename DIST, typename T, typename RES>
void run_ccn_shuffle_multimat_right_work_distribution(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread,
    hipStream_t hip_stream
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Too many warps per thread block: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (right_matrices_per_thread > right_matrices_per_thread_limit) {
        throw std::runtime_error("Too many right matrices per thread: "s +
                                 std::to_string(right_matrices_per_thread) +
                                 " (max "s +
                                 std::to_string(right_matrices_per_thread_limit) +
                                 ")"s
        );
    }

    ccn_shuffle_multimat_right_work_distribution_right_mats_dispatch<right_matrices_per_thread_limit, DIST>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_right_matrices,
        warps_per_thread_block,
        right_matrices_per_thread,
        max_rows_per_thread,
        hip_stream
    );
}

// template void run_ccn_shuffle_multimat_right<int, int>(
//         const int* __restrict__ left,
//         const int* __restrict__ right,
//         int* __restrict__ out,
//         dsize2_t matrix_size,
//         dsize2_t search_size,
//         dsize_t num_right_matrices,
//         dsize_t warps_per_thread_block,
//         dsize_t right_matrices_per_thread
// );

template void run_ccn_shuffle_multimat_right<float, float>(
        const float* __restrict__ left,
        const float* __restrict__ right,
        float* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t num_right_matrices,
        dsize_t warps_per_thread_block,
        dsize_t right_matrices_per_thread
);

// template void run_ccn_shuffle_multimat_right<double, double>(
//         const double* __restrict__ left,
//         const double* __restrict__ right,
//         double* __restrict__ out,
//         dsize2_t matrix_size,
//         dsize2_t search_size,
//         dsize_t num_right_matrices,
//         dsize_t warps_per_thread_block,
//         dsize_t right_matrices_per_thread
// );

// template void run_ccn_shuffle_multimat_right_work_distribution<triangle_distribution, int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

template void run_ccn_shuffle_multimat_right_work_distribution<triangle_distribution, float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread,
    hipStream_t hip_stream
);

// template void run_ccn_shuffle_multimat_right_work_distribution<triangle_distribution, double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

// template void run_ccn_shuffle_multimat_right_work_distribution<rectangle_distribution, int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

template void run_ccn_shuffle_multimat_right_work_distribution<rectangle_distribution, float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread,
    hipStream_t hip_stream
);

// template void run_ccn_shuffle_multimat_right_work_distribution<rectangle_distribution, double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

// template void run_ccn_shuffle_multimat_right_work_distribution<no_distribution, int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

template void run_ccn_shuffle_multimat_right_work_distribution<no_distribution, float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t max_rows_per_thread,
    hipStream_t hip_stream
);

// template void run_ccn_shuffle_multimat_right_work_distribution<no_distribution, double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_matrices_per_thread,
//     dsize_t max_rows_per_thread,
//     hipStream_t hip_stream
// );

}
