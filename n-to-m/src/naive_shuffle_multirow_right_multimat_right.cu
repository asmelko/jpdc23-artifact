#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

constexpr dsize_t right_rows_limit = SHUFFLE_MULTIROW_RIGHT_MULTIMAT_RIGHT_RIGHT_ROWS_LIMIT;
constexpr dsize_t right_mats_limit = SHUFFLE_MULTIROW_RIGHT_MULTIMAT_RIGHT_RIGHT_MATS_LIMIT;

/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
        output_pos(output_pos), matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t SUM_START, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES (&sum)[MAX_NUM_RIGHT_ROWS * NUM_RIGHT_MATS]
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

    // Preload the first values from left matrix
    T thread_left_bottom = load_with_bounds_check(
        left_row,
        warp_x_left + warp.thread_rank(),
        args.matrix_size.x
    );

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
    ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        T thread_right[NUM_RIGHT_ROWS * NUM_RIGHT_MATS];
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                thread_right[mat * NUM_RIGHT_ROWS + row] = load_with_bounds_check(
                    first_right_row + mat * args.matrix_size.area() + row * args.matrix_size.x,
                    right_idx,
                    args.matrix_size.x
                );
            }
        }

        T thread_left_top = load_with_bounds_check(left_row, left_idx, args.matrix_size.x);

        for (dsize_t i = 0; i < warp.size(); ++i) {
            for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
                for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                    // Broadcast
                    auto right_val = warp.shfl(thread_right[mat * NUM_RIGHT_ROWS + row], i);

                    // As we need to offset the sum by SUM_START exluding given shifts from ALL matrices
                    // we need to order it so that first are shifts 0 from all matrices,
                    // then shifts 1 from all matrices etc.
                    // so mat MUST be the last dimension
                    // This is why the shift, computed as described below, must be multiplied by NUM_RIGHT_MATS
                    // (NUM_RIGHT_ROWS - 1 - r) as the rows from EACH right matrix are loaded top to bottom
                    // but as we compute them agains last row from the left matrix they overlap with,
                    // the row 0 from the right matrix overlaps with the given row from the left matrix
                    // in overlap NUM_RIGHT_ROWS - 1 etc.
                    //
                    // The SUM_START is provided as during wind_down step with k rows, we need only the last
                    // k overlaps, not the first k
                    sum[SUM_START + (NUM_RIGHT_ROWS - 1 - row) * NUM_RIGHT_MATS + mat] += thread_left_bottom * right_val;
                }
            }
            // Shuffle does modulo srcLane automatically
            // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
            //  making it behave as one continuous buffer
            thread_left_bottom = warp.shfl(
                warp.thread_rank() != 0 ? thread_left_bottom : thread_left_top,
                warp.thread_rank() + 1
            );
            thread_left_top = warp.shfl_down(thread_left_top, 1);
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_NUM_RIGHT_ROWS * NUM_RIGHT_MATS]
) {
    if constexpr(NUM_RIGHT_ROWS < MAX_NUM_RIGHT_ROWS) {
        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_RIGHT_ROWS - 1 >= 0) {
            compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, MAX_NUM_RIGHT_ROWS, 0>(
                warp,
                args,
                args.warp_right_start.y,
                args.warp_min_shift.y + NUM_RIGHT_ROWS - 1,
                sum
            );
        }
        startup<NUM_RIGHT_ROWS + 1, MAX_NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<int NUM_RIGHT_ROWS, dsize_t MAX_NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_NUM_RIGHT_ROWS * NUM_RIGHT_MATS]
) {
    if constexpr(NUM_RIGHT_ROWS > 0) {
        if (args.warp_right_end.y - NUM_RIGHT_ROWS + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, MAX_NUM_RIGHT_ROWS, (MAX_NUM_RIGHT_ROWS - NUM_RIGHT_ROWS) * NUM_RIGHT_MATS>(
                warp,
                args,
                args.warp_right_end.y - NUM_RIGHT_ROWS,
                args.warp_max_shift.y,
                sum
            );
        }
        wind_down<NUM_RIGHT_ROWS - 1, MAX_NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_multirow_right_multimat_right_impl(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args
) {
    T sum[NUM_RIGHT_ROWS * NUM_RIGHT_MATS];
    for (dsize_t r = 0; r < NUM_RIGHT_ROWS * NUM_RIGHT_MATS; ++r) {
        sum[r] = 0;
    }

    startup<1, NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(warp, args, sum);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     *
     * As we are always loading warp_y_right and the following (NUM_RIGHT_ROWS - 1) rows,
     * we need to stop NUM_RIGHT_ROWS before the end
     */
    int end = args.warp_right_end.y - (NUM_RIGHT_ROWS - 1);

    for (int warp_y_right = args.warp_right_start.y; warp_y_right < end; warp_y_right += 1) {
        compute_row_group<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, NUM_RIGHT_ROWS, 0>(
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            sum
        );
    }

    wind_down<NUM_RIGHT_ROWS - 1, NUM_RIGHT_ROWS, NUM_RIGHT_MATS>(warp, args, sum);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                auto output_offset = first_output_offset + mat * args.search_size.area() + row * args.search_size.x;

                // Sum is ordered first shift 0 from all mats, then shift 1 from all mats etc.
                // as we need to exclude given shifts from all mats in wind_down using the offset
                auto val = sum[row * NUM_RIGHT_MATS + mat];
                if constexpr(ATOMIC) {
                    atomicAdd(matrix + output_offset, val);
                } else {
                    matrix[output_offset] = val;
                }
            }
        }
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_multirow_right_multimat_right_impl_mats_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_RIGHT_MATS == 0) {
        // Silence the unused parameter warning
        (void)warp;
        (void)num_right_mats;
        (void)args;
        assert(false);
    } else {
        if (NUM_RIGHT_MATS == num_right_mats) {
            shuffle_multirow_right_multimat_right_impl<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, ATOMIC>(
                warp,
                args
            );
        } else {
            shuffle_multirow_right_multimat_right_impl_mats_dispatch<NUM_RIGHT_ROWS, NUM_RIGHT_MATS - 1, ATOMIC>(
                warp,
                num_right_mats,
                args
            );
        }
    }
}

template<dsize_t NUM_RIGHT_ROWS, dsize_t NUM_RIGHT_MATS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_multirow_right_multimat_right_impl_rows_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_rows,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_RIGHT_ROWS == 0) {
        // Zero is valid, if the warp is completely outside the result matrix

        // Silence the unused parameter warning
        (void)warp;
        (void)num_right_rows;
        (void)num_right_mats;
        (void)args;
    } else {
        if (NUM_RIGHT_ROWS == num_right_rows) {
            shuffle_multirow_right_multimat_right_impl_mats_dispatch<NUM_RIGHT_ROWS, NUM_RIGHT_MATS, ATOMIC>(
                warp,
                num_right_mats,
                args
            );
        } else {
            shuffle_multirow_right_multimat_right_impl_rows_dispatch<NUM_RIGHT_ROWS - 1, NUM_RIGHT_MATS, ATOMIC>(
                warp,
                num_right_rows,
                num_right_mats,
                args
            );
        }
    }
}


template<dsize_t MAX_RIGHT_ROWS_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename T, typename RES>
__global__ void ccn_shuffle_multirow_right_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices
) {

    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    dsize_t blocks_per_matrix_group = div_up(search_size.x, warp_size);
    dsize_t matrix_group_idx = ctb.group_index().x / blocks_per_matrix_group;
    dsize_t matrix_group_block_offset = ctb.group_index().x % blocks_per_matrix_group;

    dsize_t output_x_offset = matrix_group_block_offset * warp_size;
    dsize_t matrix_group_start_idx = matrix_group_idx * MAX_RIGHT_MATRICES_PER_THREAD;

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        output_x_offset,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * MAX_RIGHT_ROWS_PER_THREAD
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift{
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift{
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) - static_cast<int>(half_search_size.x),
        // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
        static_cast<int>(min(last_warp_thread_out_pos.y + MAX_RIGHT_ROWS_PER_THREAD - 1, search_size.y - 1)) -
        static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto warp_num_right_rows = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));

    dsize_t warp_num_right_matrices = min(num_right_matrices - matrix_group_start_idx, MAX_RIGHT_MATRICES_PER_THREAD);


    auto args = create_warp_shuffle_impl_args(
        left,
        right + matrix_group_start_idx * matrix_size.area(),
        out + matrix_group_start_idx * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    shuffle_multirow_right_multimat_right_impl_rows_dispatch<MAX_RIGHT_ROWS_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD, false>(
        warp,
        warp_num_right_rows,
        warp_num_right_matrices,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_multirow_right_multimat_right_kernel_args : public kernel_args {
public:
    dsize_t max_right_rows_per_thread_;
    dsize_t max_right_matrices_per_thread_;

    ccn_shuffle_multirow_right_multimat_right_kernel_args(const ccn_shuffle_multirow_right_multimat_right_kernel_args&) = delete;
    ccn_shuffle_multirow_right_multimat_right_kernel_args& operator=(ccn_shuffle_multirow_right_multimat_right_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_right_rows_per_thread,
        dsize_t max_right_matrices_per_thread
    ) {
        static ccn_shuffle_multirow_right_multimat_right_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_right_rows_per_thread_ = max_right_rows_per_thread;
        instance.max_right_matrices_per_thread_ = max_right_matrices_per_thread;
        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_right_rows_per_thread", std::to_string(max_right_rows_per_thread_)},
            {"max_right_matrices_per_thread", std::to_string(max_right_matrices_per_thread_)}
        };
    }

private:
    ccn_shuffle_multirow_right_multimat_right_kernel_args()
        : kernel_args(),
          max_right_rows_per_thread_(0),
          max_right_matrices_per_thread_(0)
    { }
};

template<dsize_t MAX_RIGHT_ROWS_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename T, typename RES>
__host__ void ccn_shuffle_multirow_right_multimat_right_mat_disptach(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    hipStream_t hip_stream
) {
    if constexpr(MAX_RIGHT_MATRICES_PER_THREAD > 0) {
        if (MAX_RIGHT_MATRICES_PER_THREAD == right_matrices_per_thread) {
            dim3 num_threads(warp_size, warps_per_thread_block);

            dsize_t num_matrix_groups = div_up(num_right_matrices, MAX_RIGHT_MATRICES_PER_THREAD);
            dsize_t blocks_per_matrix_group = div_up(search_size.x, num_threads.x);

            dim3 num_blocks(
                blocks_per_matrix_group * num_matrix_groups,
                div_up(search_size.y, num_threads.y * MAX_RIGHT_ROWS_PER_THREAD)
            );

            ccn_shuffle_multirow_right_multimat_right<MAX_RIGHT_ROWS_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD><<<num_blocks, num_threads, 0, hip_stream>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices
            );

            ccn_shuffle_multirow_right_multimat_right_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_RIGHT_ROWS_PER_THREAD,
                MAX_RIGHT_MATRICES_PER_THREAD
            );
        } else {
            ccn_shuffle_multirow_right_multimat_right_mat_disptach<MAX_RIGHT_ROWS_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD - 1>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                hip_stream
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_matrices_per_thread;
        (void)hip_stream;
        assert(false);
    }
}

template<dsize_t MAX_RIGHT_ROWS_PER_THREAD, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, typename T, typename RES>
__host__ void ccn_shuffle_multirow_right_multimat_right_rows_disptach(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_rows_per_thread,
    dsize_t right_matrices_per_thread,
    hipStream_t hip_stream
) {
    if constexpr(MAX_RIGHT_ROWS_PER_THREAD > 0) {
        if (MAX_RIGHT_ROWS_PER_THREAD == right_rows_per_thread) {
            ccn_shuffle_multirow_right_multimat_right_mat_disptach<MAX_RIGHT_ROWS_PER_THREAD, MAX_RIGHT_MATRICES_PER_THREAD>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                hip_stream
            );
        } else {
            ccn_shuffle_multirow_right_multimat_right_rows_disptach<MAX_RIGHT_ROWS_PER_THREAD - 1, MAX_RIGHT_MATRICES_PER_THREAD>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_rows_per_thread,
                right_matrices_per_thread,
                hip_stream
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_rows_per_thread;
        (void)right_matrices_per_thread;
        (void)hip_stream;
        assert(false);
    }
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_shuffle_multirow_right_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_rows_per_thread,
    dsize_t right_matrices_per_thread,
    hipStream_t hip_stream = nullptr
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Too many warps per thread block: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (right_rows_per_thread == 0 || right_rows_per_thread > right_rows_limit) {
        throw std::runtime_error("Invalid number of right rows per thread: "s +
                                 std::to_string(right_matrices_per_thread) +
                                 " [1-"s +
                                 std::to_string(right_rows_limit) +
                                 "]"s
        );
    }

    if (right_matrices_per_thread == 0 || right_matrices_per_thread > right_mats_limit) {
        throw std::runtime_error("Invalid number of right matrices per thread: "s +
                                 std::to_string(right_matrices_per_thread) +
                                 " [1-"s +
                                 std::to_string(right_mats_limit) +
                                 "]"s
        );
    }

    ccn_shuffle_multirow_right_multimat_right_rows_disptach<right_rows_limit, right_mats_limit>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_right_matrices,
        warps_per_thread_block,
        right_rows_per_thread,
        right_matrices_per_thread,
        hip_stream
    );
}

// template void run_ccn_shuffle_multirow_right_multimat_right<int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_rows_per_thread,
//     dsize_t right_matrices_per_thread,
//     hipStream_t hip_stream
// );

template void run_ccn_shuffle_multirow_right_multimat_right<float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_rows_per_thread,
    dsize_t right_matrices_per_thread,
    hipStream_t hip_stream
);

// template void run_ccn_shuffle_multirow_right_multimat_right<double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t right_rows_per_thread,
//     dsize_t right_matrices_per_thread,
//     hipStream_t hip_stream
// );

}
