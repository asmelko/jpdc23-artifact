#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

/**
 * This kernel is a reimplementation of the original naive cross_corr kernel
 * The kernel receives reference subregions, each in row major order all stacked one after another
 * into a single array "ref". "deformed" contains corresponding subregions from "batch_size" of the deformed  pictures
 * which are to be cross-correlated with the reference subregions. All subregions are in row major order, first
 * all subregions of the first deformed image, then all subregions of the second deformed image up to the "batch_size"th
 * deformed image. Number of subregions from the reference and all the deformed images is the same.
 * The input arrays ref and deformed contain only the subregions themselfs, and we must
 * clamp the computation to use only the overlapping parts.
 *
 * For each subregion we search an area of the size "search_size" for cross-correlation maximum.
 * The whole strip of deformed subregions is partitioned into a 16x16 CUDA blocks,
 * where each thread computes one possible shift of the reference image.
 * Output contains an an array of "search_size" results in row major order
 * corresponding to the result of cross correlation for each position in the search area.
 *
 * The memory access patterns are not ideal. Due to the 16x16 size of each block,
 * each half of the warp accesses different row of the "picture", most likely leading to two 128 byte
 * global memory accesses. The implementation also does not use shared memory in any way.
 */
template<typename T, typename RES>
__global__ void cross_corr_naive_original(
    const T* __restrict__ ref,
    const T* __restrict__ deformed,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size

) {
    cg::thread_block ctb = cg::this_thread_block();

    // Coordinates in the whole strip of deformed subregions
    unsigned int def_strip_x = ctb.group_index().x * ctb.group_dim().x + ctb.thread_index().x;
    unsigned int def_strip_y = ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y;

    unsigned int region_idx = def_strip_x / search_size.x;

    if (region_idx >= subregions_per_pic || def_strip_y >= search_size.y) {
        return;
    }

    // Position of the centre of the subregion
    dsize2_t in_region_pos{def_strip_x % search_size.x, def_strip_y};
    dsize_t ref_idx = region_idx % subregions_per_pic;
    dsize2_t half_size = (search_size - 1) / 2;

    vec2<int> shift{(int)in_region_pos.x - (int)half_size.x, (int)in_region_pos.y - (int)half_size.y};

    ref += ref_idx * subregion_size.area();
    deformed += region_idx * subregion_size.area();
    out += region_idx * search_size.area();

    for (dsize_t i = 0; i < batch_size; ++i) {
        // The code is different from the original as here we are sliding the
        // deformed region over the reference region, whereas the original
        // did it the other way, which is incorrect in my opinion
        // or at least inconsistent with the text of the thesis
        // where it is defined as reference * deformed
        // and the algorithm clearly states that this means sliding the deformed
        //
        // The results also now match the results of matlab xcorr2
        dsize_t x_ref_start = max(shift.x, 0);
        dsize_t x_ref_end = min(subregion_size.x + shift.x, subregion_size.x);
        dsize_t y_ref_start = max(shift.y, 0);
        dsize_t y_ref_end = min(subregion_size.y + shift.y, subregion_size.y);

        RES sum = 0;
        for (dsize_t y_ref = y_ref_start; y_ref < y_ref_end; ++y_ref) {
            for (dsize_t x_ref = x_ref_start; x_ref < x_ref_end; ++x_ref) {
                // If deformed is shifted by -10, the we are starting from [0,0] in ref
                // and need to start from [10,10] in deformed, as there are 10
                // values to the left and on top outside the reference matrix
                int x_shifted = x_ref - shift.x;
                int y_shifted = y_ref - shift.y;

                sum += deformed[y_shifted * subregion_size.x + x_shifted] * ref[y_ref * subregion_size.x + x_ref];
            }
        }

        out[in_region_pos.linear_idx(search_size.x)] = sum;

        deformed += subregions_per_pic * subregion_size.area();
        out += subregions_per_pic * search_size.area();
    }
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class cross_corr_naive_original_kernel_args : public kernel_args {
public:
    cross_corr_naive_original_kernel_args(const cross_corr_naive_original_kernel_args&) = delete;
    cross_corr_naive_original_kernel_args& operator=(cross_corr_naive_original_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size
    ) {
        static cross_corr_naive_original_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        set_last_kernel_launch_args(&instance);
    }

private:
    cross_corr_naive_original_kernel_args()
        : kernel_args()
    { }
};

} // END anonymous namespace

template<typename T, typename RES>
void run_cross_corr_naive_original(
    const T* __restrict__ ref,
    const T* __restrict__ deformed,
    RES* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size,
    hipStream_t hip_stream = nullptr
) {
    dim3 num_threads(16, 16);
    dim3 num_blocks(
        div_up(search_size.x * subregions_per_pic, num_threads.x),
        div_up(search_size.y, num_threads.y)
    );

    cross_corr_naive_original<<<num_blocks, num_threads, 0, hip_stream>>>(
        ref,
        deformed,
        out,
        subregion_size,
        search_size,
        subregions_per_pic,
        batch_size
    );

    cross_corr_naive_original_kernel_args::record_launch(
        num_threads,
        num_blocks
    );
}

// template void run_cross_corr_naive_original<int, int>(
//     const int* __restrict__ ref,
//     const int* __restrict__ deformed,
//     int* __restrict__ out,
//     dsize2_t subregion_size,
//     dsize2_t search_size,
//     dsize_t subregions_per_pic,
//     dsize_t batch_size,
//     hipStream_t hip_stream
// );

template void run_cross_corr_naive_original<float, float>(
    const float* __restrict__ ref,
    const float* __restrict__ deformed,
    float* __restrict__ out,
    dsize2_t subregion_size,
    dsize2_t search_size,
    dsize_t subregions_per_pic,
    dsize_t batch_size,
    hipStream_t hip_stream
);

// template void run_cross_corr_naive_original<double, double>(
//     const double* __restrict__ ref,
//     const double* __restrict__ deformed,
//     double* __restrict__ out,
//     dsize2_t subregion_size,
//     dsize2_t search_size,
//     dsize_t subregions_per_pic,
//     dsize_t batch_size,
//     hipStream_t hip_stream
// );

}
