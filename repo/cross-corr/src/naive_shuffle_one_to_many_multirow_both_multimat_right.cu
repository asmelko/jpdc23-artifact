#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

constexpr dsize_t shifts_per_thread_per_right_matrix_limit = SHUFFLE_ONE_TO_MANY_MULTIROW_BOTH_MULTIMAT_RIGHT_SHIFTS_PER_RIGHT_MATRIX_LIMIT;
constexpr dsize_t right_matrices_per_thread_limit = SHUFFLE_ONE_TO_MANY_MULTIROW_BOTH_MULTIMAT_RIGHT_RIGHT_MATRICES_PER_THREAD_LIMIT;
constexpr dsize_t left_rows_per_iteration_limit = SHUFFLE_ONE_TO_MANY_MULTIROW_BOTH_MULTIMAT_RIGHT_LEFT_ROWS_PER_ITERATION_LIMIT;
/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
        output_pos(output_pos), matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_SHIFTS_PER_MAT, dsize_t NUM_RIGHT_MATS, dsize_t NUM_LEFT_ROWS, dsize_t MAX_NUM_SHIFTS_PER_MATS, dsize_t SUM_START, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES (&sum)[MAX_NUM_SHIFTS_PER_MATS * NUM_RIGHT_MATS]
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* first_left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

    // Preload the first values from left matrix
    T thread_left_bottom[NUM_LEFT_ROWS];
    #pragma unroll
    for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
        thread_left_bottom[l] = load_with_bounds_check(
            first_left_row + l * args.matrix_size.x,
            warp_x_left + warp.thread_rank(),
            args.matrix_size.x
        );
    }

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
    ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        // We need this many right values because first left row
        // is computed with rows 0 to NUM_SHIFTS_PER_MAT - 1, second left row is computed
        // with rows 1 to NUM_SHIFTS_PER_MAT, third left row with 2 to NUM_SHIFTS_PER_MAT + 1
        constexpr dsize_t NUM_RIGHT_ROWS = NUM_SHIFTS_PER_MAT + NUM_LEFT_ROWS - 1;
        constexpr dsize_t TOTAL_RIGHT_ROWS = NUM_RIGHT_ROWS * NUM_RIGHT_MATS;
        // Load values from num_rights right matrices
        T thread_right[TOTAL_RIGHT_ROWS];
        #pragma unroll
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            #pragma unroll
            for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                thread_right[mat * NUM_RIGHT_ROWS + row] = load_with_bounds_check(
                    first_right_row + mat * args.matrix_size.area() + row * args.matrix_size.x,
                    right_idx,
                    args.matrix_size.x
                );
            }
        }

        T thread_left_top[NUM_LEFT_ROWS];
        #pragma unroll
        for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
            thread_left_top[l] = load_with_bounds_check(
                first_left_row + l * args.matrix_size.x,
                left_idx,
                args.matrix_size.x
            );
        }

        // TODO: Maybe pragma unroll?
        for (dsize_t i = 0; i < warp.size(); ++i) {
            #pragma unroll
            for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
                #pragma unroll
                for (dsize_t row = 0; row < NUM_RIGHT_ROWS; ++row) {
                    // Broadcast
                    auto right_val = warp.shfl(thread_right[mat * NUM_RIGHT_ROWS + row], i);

                    #pragma unroll
                    for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
                        // Some combinations are not valid, as described by the NUM_RIGHT_ROWS
                        // variable comment.
                        // left row 0 is computed with right rows 0 to NUM_SHIFTS_PER_MAT - 1
                        // left row 1 is computed with right rows 1 to NUM_SHIFTS_PER_MAT
                        // left row 2 is computed with right rows 2 to NUM_SHIFTS_PER_MAT + 1
                        if (l <= row && row < NUM_SHIFTS_PER_MAT + l) {
                            // As we need to offset the sum by SUM_START exluding given shifts from ALL matrices
                            // we need to order it so that first are shifts 0 from all matrices,
                            // then shifts 1 from all matrices etc.
                            // so mat MUST be the last dimension
                            // This is why the shift, computed as described below, must be multiplied by NUM_RIGHT_MATS
                            // (row - l) gets us the right row index compared to the left row
                            // (NUM_SHIFTS_PER_MAT - 1 - (r - l)) as the rows from the right matrix are loaded top to bottom
                            // but as we compute them agains last row from the left matrix they overlap with,
                            // the row 0 from the right matrix overlaps with the given row from the left matrix
                            // in overlap NUM_RIGHT_ROWS - 1 etc.
                            sum[SUM_START + (NUM_SHIFTS_PER_MAT - 1 - (row - l)) * NUM_RIGHT_MATS + mat] += thread_left_bottom[l] * right_val;
                        }
                    }
                }
            }

            #pragma unroll
            for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {

                // This if cannot be changed into ternary operator
                // as nvcc fails to optimize the two arrays into registers
                // and instead puts them into local memory when ternary operator
                // is used
                T bottom_shift_val;
                if (warp.thread_rank() != 0) {
                    bottom_shift_val = thread_left_bottom[l];
                } else {
                    // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                    //  making it behave as one continuous buffer
                    bottom_shift_val = thread_left_top[l];
                }
                // Shuffle does modulo srcLane automatically
                thread_left_bottom[l] = warp.shfl(bottom_shift_val, warp.thread_rank() + 1);

                thread_left_top[l] = warp.shfl_down(thread_left_top[l], 1);
            }
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int NUM_SHIFTS_PER_MAT, dsize_t MAX_NUM_SHIFTS_PER_MATS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_NUM_SHIFTS_PER_MATS * NUM_RIGHT_MATS]
) {
    if constexpr(NUM_SHIFTS_PER_MAT < MAX_NUM_SHIFTS_PER_MATS) {
        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + NUM_SHIFTS_PER_MAT - 1 >= 0) {
            compute_row_group<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, 1, MAX_NUM_SHIFTS_PER_MATS, 0>(
                warp,
                args,
                args.warp_right_start.y,
                args.warp_min_shift.y + NUM_SHIFTS_PER_MAT - 1,
                sum
            );
        }
        startup<NUM_SHIFTS_PER_MAT + 1, MAX_NUM_SHIFTS_PER_MATS, NUM_RIGHT_MATS>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<int NUM_SHIFTS_PER_MAT, dsize_t MAX_NUM_SHIFTS_PER_MATS, dsize_t NUM_RIGHT_MATS, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_NUM_SHIFTS_PER_MATS * NUM_RIGHT_MATS]
) {
    if constexpr(NUM_SHIFTS_PER_MAT > 0) {
        if (args.warp_right_end.y - NUM_SHIFTS_PER_MAT + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, 1, MAX_NUM_SHIFTS_PER_MATS, (MAX_NUM_SHIFTS_PER_MATS - NUM_SHIFTS_PER_MAT) * NUM_RIGHT_MATS>(
                warp,
                args,
                args.warp_right_end.y - NUM_SHIFTS_PER_MAT,
                args.warp_max_shift.y,
                sum
            );
        }
        wind_down<NUM_SHIFTS_PER_MAT - 1, MAX_NUM_SHIFTS_PER_MATS, NUM_RIGHT_MATS>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<dsize_t NUM_SHIFTS_PER_MAT, dsize_t NUM_RIGHT_MATS, dsize_t LEFT_ROWS_PER_ITER, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_one_to_many_multirow_both_multimat_right_impl(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args
) {
    T sum[NUM_SHIFTS_PER_MAT * NUM_RIGHT_MATS];
    for (dsize_t s = 0; s < NUM_SHIFTS_PER_MAT * NUM_RIGHT_MATS; ++s) {
        sum[s] = 0;
    }
    startup<1, NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS>(warp, args, sum);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1 before the end
     */
    int multileft_end = args.warp_right_end.y - (NUM_SHIFTS_PER_MAT + LEFT_ROWS_PER_ITER - 1);
    int warp_y_right = args.warp_right_start.y;
    for (; warp_y_right < multileft_end; warp_y_right += LEFT_ROWS_PER_ITER) {
        compute_row_group<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, LEFT_ROWS_PER_ITER, NUM_SHIFTS_PER_MAT, 0>(
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            sum
        );
    }

    /*
     * Finish the possible MAX_LEFT_ROWS - 1 left rows left over before the original wind-down
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS before the end
     * TODO: Try template generated if tree that will use just one call with the correct number of left rows
     */
    int total_end = args.warp_right_end.y - (NUM_SHIFTS_PER_MAT - 1);
    for (; warp_y_right < total_end; warp_y_right += 1) {
        compute_row_group<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, 1, NUM_SHIFTS_PER_MAT, 0>(
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            sum
        );
    }

    wind_down<NUM_SHIFTS_PER_MAT - 1, NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS>(warp, args, sum);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        for (dsize_t mat = 0; mat < NUM_RIGHT_MATS; ++mat) {
            for (dsize_t shift = 0; shift < NUM_SHIFTS_PER_MAT; ++shift) {
                auto output_offset = first_output_offset + mat * args.search_size.area() + shift * args.search_size.x;

                // Sum is ordered first shift 0 from all mats, then shift 1 from all mats etc.
                // as we need to exclude given shifts from all mats in wind_down using the offset
                auto val = sum[shift * NUM_RIGHT_MATS + mat];
                if constexpr(ATOMIC) {
                    atomicAdd(matrix + output_offset, val);
                } else {
                    matrix[output_offset] = val;
                }
            }
        }
    }
}

template<dsize_t NUM_SHIFTS_PER_MAT, dsize_t NUM_RIGHT_MATS, dsize_t LEFT_ROWS_PER_ITER, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_one_to_many_multirow_both_multimat_right_impl_mats_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_RIGHT_MATS == 0) {
        // Silence the unused parameter warning
        (void)warp;
        (void)num_right_mats;
        (void)args;
        assert(false);
    } else {
        if (NUM_RIGHT_MATS == num_right_mats) {
            shuffle_one_to_many_multirow_both_multimat_right_impl<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, LEFT_ROWS_PER_ITER, ATOMIC>(
                warp,
                args
            );
        } else {
            shuffle_one_to_many_multirow_both_multimat_right_impl_mats_dispatch<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS - 1, LEFT_ROWS_PER_ITER, ATOMIC>(
                warp,
                num_right_mats,
                args
            );
        }
    }
}

template<dsize_t NUM_SHIFTS_PER_MAT, dsize_t NUM_RIGHT_MATS, dsize_t LEFT_ROWS_PER_ITER, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_one_to_many_multirow_both_multimat_right_impl_shifts_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_shifts_per_mat,
    dsize_t num_right_mats,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(NUM_SHIFTS_PER_MAT == 0) {
        // Zero is valid, if the warp is completely outside the result matrix

        // Silence the unused parameter warning
        (void)warp;
        (void)num_shifts_per_mat;
        (void)num_right_mats;
        (void)args;
    } else {
        if (NUM_SHIFTS_PER_MAT == num_shifts_per_mat) {
            shuffle_one_to_many_multirow_both_multimat_right_impl_mats_dispatch<NUM_SHIFTS_PER_MAT, NUM_RIGHT_MATS, LEFT_ROWS_PER_ITER, ATOMIC>(
                warp,
                num_right_mats,
                args
            );
        } else {
            shuffle_one_to_many_multirow_both_multimat_right_impl_shifts_dispatch<NUM_SHIFTS_PER_MAT - 1,  NUM_RIGHT_MATS, LEFT_ROWS_PER_ITER, ATOMIC>(
                warp,
                num_shifts_per_mat,
                num_right_mats,
                args
            );
        }
    }
}


/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<dsize_t MAX_SHIFTS_PER_RIGHT_MATRIX, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, dsize_t LEFT_ROWS_PER_ITER, typename T, typename RES>
__global__ void ccn_shuffle_one_to_many_multirow_both_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices
) {
    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    dsize_t blocks_per_matrix_group = div_up(search_size.x, warp_size);
    dsize_t matrix_group_idx = ctb.group_index().x / blocks_per_matrix_group;
    dsize_t matrix_group_block_offset = ctb.group_index().x % blocks_per_matrix_group;

    dsize_t output_x_offset = matrix_group_block_offset * warp_size;
    dsize_t matrix_group_start_idx = matrix_group_idx * MAX_RIGHT_MATRICES_PER_THREAD;

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        output_x_offset,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * MAX_SHIFTS_PER_RIGHT_MATRIX
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) -
        static_cast<int>(half_search_size.x),
        // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
        static_cast<int>(min(last_warp_thread_out_pos.y + MAX_SHIFTS_PER_RIGHT_MATRIX - 1, search_size.y - 1)) -
        static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto warp_num_shifts_per_right_mat = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));
    dsize_t warp_num_right_matrices = min(num_right_matrices - matrix_group_start_idx, MAX_RIGHT_MATRICES_PER_THREAD);

    auto args = create_warp_shuffle_impl_args(
        left,
        right + matrix_group_start_idx * matrix_size.area(),
        out + matrix_group_start_idx * search_size.area(),
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    shuffle_one_to_many_multirow_both_multimat_right_impl_shifts_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER, false>(
        warp,
        warp_num_shifts_per_right_mat,
        warp_num_right_matrices,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args : public kernel_args {
public:
    dsize_t max_shifts_per_right_matrix_;
    dsize_t max_right_matrices_per_thread_;
    dsize_t left_rows_per_iter_;

    ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args(const ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args&) = delete;
    ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args& operator=(ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_shifts_per_right_matrix,
        dsize_t max_right_matrices_per_thread,
        dsize_t left_rows_per_iter
    ) {
        static ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_shifts_per_right_matrix_ = max_shifts_per_right_matrix;
        instance.max_right_matrices_per_thread_ = max_right_matrices_per_thread;
        instance.left_rows_per_iter_ = left_rows_per_iter;
        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_shifts_per_right_matrix", std::to_string(max_shifts_per_right_matrix_)},
            {"max_right_matrices_per_thread", std::to_string(max_right_matrices_per_thread_)},
            {"left_rows_per_iter", std::to_string(left_rows_per_iter_)}
        };
    }

private:
    ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args()
        : kernel_args(),
          max_shifts_per_right_matrix_(0),
          max_right_matrices_per_thread_(0),
          left_rows_per_iter_(0)
    { }
};

template<dsize_t MAX_SHIFTS_PER_RIGHT_MATRIX, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, dsize_t LEFT_ROWS_PER_ITER, typename T, typename RES>
__host__ void ccn_shuffle_one_to_many_multirow_both_multimat_right_left_rows_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t left_rows_per_iteration,
    hipStream_t hip_stream
) {
    if constexpr(LEFT_ROWS_PER_ITER > 0) {
        if (LEFT_ROWS_PER_ITER == left_rows_per_iteration) {
            dim3 num_threads(warp_size, warps_per_thread_block);

            dsize_t num_matrix_groups = div_up(num_right_matrices, MAX_RIGHT_MATRICES_PER_THREAD);
            dsize_t blocks_per_matrix_group = div_up(search_size.x, num_threads.x);

            dim3 num_blocks(
                blocks_per_matrix_group * num_matrix_groups,
                div_up(search_size.y, num_threads.y * MAX_SHIFTS_PER_RIGHT_MATRIX)
            );

            ccn_shuffle_one_to_many_multirow_both_multimat_right<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER><<<num_blocks, num_threads, 0, hip_stream>>>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices
            );

            ccn_shuffle_one_to_many_multirow_both_multimat_right_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_SHIFTS_PER_RIGHT_MATRIX,
                MAX_RIGHT_MATRICES_PER_THREAD,
                LEFT_ROWS_PER_ITER
            );

        } else {
            ccn_shuffle_one_to_many_multirow_both_multimat_right_left_rows_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER - 1>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                left_rows_per_iteration,
                hip_stream
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)left_rows_per_iteration;
        (void)hip_stream;
        assert(false);
    }
}

template<dsize_t MAX_SHIFTS_PER_RIGHT_MATRIX, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, dsize_t LEFT_ROWS_PER_ITER, typename T, typename RES>
__host__ void ccn_shuffle_one_to_many_multirow_both_multimat_right_right_mats_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t right_matrices_per_thread,
    dsize_t left_rows_per_iteration,
    hipStream_t hip_stream
) {
    if constexpr(MAX_RIGHT_MATRICES_PER_THREAD > 0) {
        if (MAX_RIGHT_MATRICES_PER_THREAD == right_matrices_per_thread) {
            ccn_shuffle_one_to_many_multirow_both_multimat_right_left_rows_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                left_rows_per_iteration,
                hip_stream
            );
        } else {
            ccn_shuffle_one_to_many_multirow_both_multimat_right_right_mats_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD - 1, LEFT_ROWS_PER_ITER>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                left_rows_per_iteration,
                hip_stream
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)right_matrices_per_thread;
        (void)left_rows_per_iteration;
        (void)hip_stream;
        assert(false);
    }
}

template<dsize_t MAX_SHIFTS_PER_RIGHT_MATRIX, dsize_t MAX_RIGHT_MATRICES_PER_THREAD, dsize_t LEFT_ROWS_PER_ITER, typename T, typename RES>
__host__ void ccn_shuffle_one_to_many_multirow_both_multimat_right_shifts_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t shifts_per_thread_right_matrix,
    dsize_t right_matrices_per_thread,
    dsize_t left_rows_per_iteration,
    hipStream_t hip_stream
) {
    if constexpr(MAX_SHIFTS_PER_RIGHT_MATRIX > 0) {
        if (MAX_SHIFTS_PER_RIGHT_MATRIX == shifts_per_thread_right_matrix) {
            ccn_shuffle_one_to_many_multirow_both_multimat_right_right_mats_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                right_matrices_per_thread,
                left_rows_per_iteration,
                hip_stream
            );
        } else {
            ccn_shuffle_one_to_many_multirow_both_multimat_right_shifts_dispatch<MAX_SHIFTS_PER_RIGHT_MATRIX - 1, MAX_RIGHT_MATRICES_PER_THREAD, LEFT_ROWS_PER_ITER>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                num_right_matrices,
                warps_per_thread_block,
                shifts_per_thread_right_matrix,
                right_matrices_per_thread,
                left_rows_per_iteration,
                hip_stream
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)num_right_matrices;
        (void)warps_per_thread_block;
        (void)shifts_per_thread_right_matrix;
        (void)right_matrices_per_thread;
        (void)left_rows_per_iteration;
        (void)hip_stream;
        assert(false);
    }
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_shuffle_one_to_many_multirow_both_multimat_right(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t shifts_per_thread_right_matrix,
    dsize_t right_matrices_per_thread,
    dsize_t left_rows_per_iteration,
    hipStream_t hip_stream = nullptr
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Cuda block too large: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (shifts_per_thread_right_matrix > shifts_per_thread_per_right_matrix_limit) {
        throw std::runtime_error(
            "Too many shifts per thread in each right matrix: "s +
            std::to_string(shifts_per_thread_right_matrix) +
            "(max "s + std::to_string(shifts_per_thread_per_right_matrix_limit) +
            ")"s
        );
    }

    if (right_matrices_per_thread == 0 || right_matrices_per_thread > right_matrices_per_thread_limit) {
        throw std::runtime_error("Invalid number of right matrices per thread: "s +
                                 std::to_string(right_matrices_per_thread) +
                                 " [1-"s +
                                 std::to_string(right_matrices_per_thread_limit) +
                                 "]"s
        );
    }

    if (left_rows_per_iteration > left_rows_per_iteration_limit) {
        throw std::runtime_error(
            "Too many left rows per iteration: "s +
            std::to_string(left_rows_per_iteration) +
            "(max "s + std::to_string(left_rows_per_iteration_limit) +
            ")"s
        );
    }

    ccn_shuffle_one_to_many_multirow_both_multimat_right_shifts_dispatch<shifts_per_thread_per_right_matrix_limit, right_matrices_per_thread_limit, left_rows_per_iteration_limit>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        num_right_matrices,
        warps_per_thread_block,
        shifts_per_thread_right_matrix,
        right_matrices_per_thread,
        left_rows_per_iteration,
        hip_stream
    );
}

// template void run_ccn_shuffle_one_to_many_multirow_both_multimat_right<int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t shifts_per_thread_right_matrix,
//     dsize_t right_matrices_per_thread,
//     dsize_t left_rows_per_iteration,
//     hipStream_t hip_stream
// );

template void run_ccn_shuffle_one_to_many_multirow_both_multimat_right<float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t num_right_matrices,
    dsize_t warps_per_thread_block,
    dsize_t shifts_per_thread_right_matrix,
    dsize_t right_matrices_per_thread,
    dsize_t left_rows_per_iteration,
    hipStream_t hip_stream
);

// template void run_ccn_shuffle_one_to_many_multirow_both_multimat_right<double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t num_right_matrices,
//     dsize_t warps_per_thread_block,
//     dsize_t shifts_per_thread_right_matrix,
//     dsize_t right_matrices_per_thread,
//     dsize_t left_rows_per_iteration,
//     hipStream_t hip_stream
// );

}
