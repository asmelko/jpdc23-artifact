#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

#include <stdexcept>
#include <cassert>

#include "types.cuh"
#include "cuda_helpers.cuh"
#include "bound_checked_loads.cuh"

#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

constexpr dsize_t shifts_per_thread_limit = SHUFFLE_MULTIROW_BOTH_SHIFTS_PER_THREAD_LIMIT;
constexpr dsize_t left_rows_limit = SHUFFLE_MULTIROW_BOTH_LEFT_ROWS_LIMIT;
/**
 * Arguments for the warp_shuffle_impl function.
 * As we need to write many calls for different constant values of NUM_RIGHTS which
 * all share the same argument values, we want to have each call as short as possible
 * This way, we can create the arguments with a single call and then use it in any of the calls in the switch statement
 *
 * @tparam T
 * @tparam RES
 */
template<typename T, typename RES>
struct warp_shuffle_impl_args {
    const T* __restrict__ left;
    const T* __restrict__ right;
    RES* __restrict__ out;
    dsize2_t warp_right_start;
    dsize2_t warp_right_end;
    vec2<int> warp_min_shift;
    vec2<int> warp_max_shift;
    dsize2_t output_pos;
    dsize2_t matrix_size;
    dsize2_t search_size;

    __device__ warp_shuffle_impl_args(
        const T* __restrict__ left,
        const T* __restrict__ right,
        RES* __restrict__ out,
        dsize2_t warp_right_start,
        dsize2_t warp_right_end,
        vec2<int> warp_min_shift,
        vec2<int> warp_max_shift,
        dsize2_t output_pos,
        dsize2_t matrix_size,
        dsize2_t search_size
    ) : left(left), right(right), out(out), warp_right_start(warp_right_start),
        warp_right_end(warp_right_end), warp_min_shift(warp_min_shift), warp_max_shift(warp_max_shift),
        output_pos(output_pos), matrix_size(matrix_size), search_size(search_size) {

    }
};

template<typename T, typename RES>
__device__ warp_shuffle_impl_args<T, RES> create_warp_shuffle_impl_args(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t warp_right_start,
    dsize2_t warp_right_end,
    vec2<int> warp_min_shift,
    vec2<int> warp_max_shift,
    dsize2_t output_pos,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    return warp_shuffle_impl_args<T, RES>(
        left,
        right,
        out,
        warp_right_start,
        warp_right_end,
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );
}

template<dsize_t NUM_SHIFTS, dsize_t NUM_LEFT_ROWS, dsize_t MAX_NUM_SHIFTS, dsize_t SUM_START, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void compute_row_group(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    dsize_t warp_y_right_start,
    int y_shift,
    RES (&sum)[MAX_NUM_SHIFTS]
) {
    dsize_t warp_y_left = warp_y_right_start + y_shift;
    const T* first_left_row = args.left + warp_y_left * args.matrix_size.x;

    const dsize_t first_right_row_offset = warp_y_right_start * args.matrix_size.x;
    const T* first_right_row = args.right + first_right_row_offset;

    int warp_x_left = static_cast<int>(args.warp_right_start.x) + args.warp_min_shift.x;

    // Preload the first values from left matrix
    T thread_left_bottom[NUM_LEFT_ROWS];
    #pragma unroll
    for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
        thread_left_bottom[l] = load_with_bounds_check(
            first_left_row + l * args.matrix_size.x,
            warp_x_left + warp.thread_rank(),
            args.matrix_size.x
        );
    }

    for (
        dsize_t warp_x_right = args.warp_right_start.x;
        warp_x_right < args.warp_right_end.x;
        warp_x_right += warp.size(), warp_x_left += warp.size()
        ) {

        // Load next warp_size values
        // Load 0 if out of bounds

        // Right index will always be greater than 0 as we only
        // iterate over part of the matrix
        dsize_t right_idx = warp_x_right + warp.thread_rank();

        // Left index might be out of bounds even below 0, depending on the shift
        // It is also reading warp.size() next values, as we have warp.size() values already loaded
        // from the initialization before the for loop
        int left_idx = warp_x_left + warp.thread_rank() + warp.size();

        // We need this many right values because first left row
        // is computed with rows 0 to NUM_SHIFTS - 1, second left row is computed
        // with rows 1 to NUM_SHIFTS, third left row with 2 to NUM_SHIFTS + 1
        constexpr dsize_t NUM_RIGHT_ROWS = NUM_SHIFTS + NUM_LEFT_ROWS - 1;
        // Load values from num_rights right matrices
        T thread_right[NUM_RIGHT_ROWS];
        #pragma unroll
        for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
            // TODO: Either do bounds check or limit the for loop below
            thread_right[r] = load_with_bounds_check(
                first_right_row + r * args.matrix_size.x,
                right_idx,
                args.matrix_size.x
            );
        }


        T thread_left_top[NUM_LEFT_ROWS];
        #pragma unroll
        for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
            thread_left_top[l] = load_with_bounds_check(
                first_left_row + l * args.matrix_size.x,
                left_idx,
                args.matrix_size.x
            );
        }

        // TODO: Maybe pragma unroll?
        for (dsize_t i = 0; i < warp.size(); ++i) {
            #pragma unroll
            for (dsize_t r = 0; r < NUM_RIGHT_ROWS; ++r) {
                // Broadcast
                auto right_val = warp.shfl(thread_right[r], i);

                #pragma unroll
                for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {
                    // Some combinations are not valid, as described by the NUM_RIGHT_ROWS
                    // variable comment.
                    // left row 0 is computed with right rows 0 to NUM_SHIFTS - 1
                    // left row 1 is computed with right rows 1 to NUM_SHIFTS
                    // left row 2 is computed with right rows 2 to NUM_SHIFTS + 1
                    // TODO: Try if using break or continue can still be unrolled
                    if (l <= r && r < NUM_SHIFTS + l) {
                        // (r - l) gets us the right row index compared to the left row
                        // (NUM_SHIFTS - 1 - (r - l)) as the rows from the right matrix are loaded top to bottom
                        // but as we compute them agains last row from the left matrix they overlap with,
                        // the row 0 from the right matrix overlaps with the given row from the left matrix
                        // in overlap NUM_RIGHT_ROWS - 1 etc.
                        //
                        // The SUM_START as during wind_down step with k rows, we need only the last
                        // k overlaps, not the first k
                        sum[SUM_START + (NUM_SHIFTS - 1 - (r - l))] += thread_left_bottom[l] * right_val;
                    }
                }
            }

            #pragma unroll
            for (dsize_t l = 0; l < NUM_LEFT_ROWS; ++l) {

                // This if cannot be changed into ternary operator
                // as nvcc fails to optimize the two arrays into registers
                // and instead puts them into local memory when ternary operator
                // is used
                T bottom_shift_val;
                if (warp.thread_rank() != 0) {
                    bottom_shift_val = thread_left_bottom[l];
                } else {
                    // Lane 0 pushes the bottom-most value of the top buffer to the top of the bottom buffer
                    //  making it behave as one continuous buffer
                    bottom_shift_val = thread_left_top[l];
                }
                // Shuffle does modulo srcLane automatically
                thread_left_bottom[l] = warp.shfl(bottom_shift_val, warp.thread_rank() + 1);

                thread_left_top[l] = warp.shfl_down(thread_left_top[l], 1);
            }
        }
    }
}

/*
 * First NUM_RIGHT_ROWS rows will only overlap in some of the shifts
 * If we start at the 0 row of the right matrix, then that means that the
 * top of the right matrix is inside the left matrix
 *
 * As we are computing NUM_RIGHT_ROWS shifts in consecutive rows with the same
 * x coordinate, the first shift will overlap given left row and no other shift
 * overlaps anything with the left row
 *
 * Next left row is overlapped with the args.warp_right_start.y by the following shift,
 * while the first shift overlaps the left row with args.warp_right_start.y + 1
 *
 * Then the third left row is overlapped with args.warp_right_start.y by the third shift,
 * with args.warp_right_start.y + 1 by second shift and with args.warp_right_start.y + 2 by
 * first shift etc.
 *
 * If the top of the right matrix starts outside the left matrix, which can only be above the
 * left matrix, some of the steps may be skipped, for example if it is one row above,
 * the first left row is overlapped by the first shift with row args.warp_right_start.y + 1
 * and by the second shift with row args.warp_right_start.y, which is exactly the second step described above
 *
 * Similar principle, but in reverse, applies when bottom of the right matrix is inside the left matrix.
 * There the left row stays the same, but we change the number of right rows it runs against,
 * getting progressively smaller.
 *
 * These ifs should cover all possibilities up to NUM_RIGHT_ROWS
 * Because max_shift.y - min_shift.y == NUM_RIGHT_ROWS, min_shift.y + NUM_RIGHT_ROWS == max_shift.y
 *
 */
template<int SHIFTS_PER_THREAD, dsize_t MAX_SHIFTS_PER_THREAD, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void startup(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_SHIFTS_PER_THREAD]
) {
    if constexpr(SHIFTS_PER_THREAD < MAX_SHIFTS_PER_THREAD) {
        if (static_cast<int>(args.warp_right_start.y) + args.warp_min_shift.y + SHIFTS_PER_THREAD - 1 >= 0) {
            compute_row_group<SHIFTS_PER_THREAD, 1, MAX_SHIFTS_PER_THREAD, 0>(
                warp,
                args,
                args.warp_right_start.y,
                args.warp_min_shift.y + SHIFTS_PER_THREAD - 1,
                sum
            );
        }
        startup<SHIFTS_PER_THREAD + 1, MAX_SHIFTS_PER_THREAD>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<int SHIFTS_PER_THREAD, dsize_t MAX_SHIFTS_PER_THREAD, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void wind_down(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args,
    RES (&sum)[MAX_SHIFTS_PER_THREAD]
) {
    if constexpr(SHIFTS_PER_THREAD > 0) {
        if (args.warp_right_end.y - SHIFTS_PER_THREAD + args.warp_max_shift.y < args.matrix_size.y) {
            compute_row_group<SHIFTS_PER_THREAD, 1, MAX_SHIFTS_PER_THREAD, MAX_SHIFTS_PER_THREAD - SHIFTS_PER_THREAD>(
                warp,
                args,
                args.warp_right_end.y - SHIFTS_PER_THREAD,
                args.warp_max_shift.y,
                sum
            );
        }
        wind_down<SHIFTS_PER_THREAD - 1, MAX_SHIFTS_PER_THREAD>(warp, args, sum);
    } else {
        // Silence the unused parameter warning
        (void)warp;
        (void)args;
        (void)sum;
    }
}

template<dsize_t SHIFTS_PER_THREAD, dsize_t MAX_LEFT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_multirow_both_impl(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    warp_shuffle_impl_args<T, RES> args
) {
    T sum[SHIFTS_PER_THREAD];
    for (dsize_t s = 0; s < SHIFTS_PER_THREAD; ++s) {
        sum[s] = 0;
    }

    startup<1, SHIFTS_PER_THREAD>(warp, args, sum);

    /*
     * The startup gets us to the situation where we have the first
     * left row at max_shift (== min_shift + NUM_RIGHTS_ROW) which is
     * to be processed with all NUM_RIGHT_ROWS
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS + MAX_LEFT_ROWS - 1 before the end
     */
    int multileft_end = args.warp_right_end.y - (SHIFTS_PER_THREAD + MAX_LEFT_ROWS - 1);
    int warp_y_right = args.warp_right_start.y;
    for (; warp_y_right < multileft_end; warp_y_right += MAX_LEFT_ROWS) {
        compute_row_group<SHIFTS_PER_THREAD, MAX_LEFT_ROWS, SHIFTS_PER_THREAD, 0>(
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            sum
        );
    }

    /*
     * Finish the possible MAX_LEFT_ROWS - 1 left rows left over before the original wind-down
     * As we are always loading warp_y_right and the following (NUM_THREAD_SHIFTS - 1) rows,
     * we need to stop NUM_THREAD_SHIFTS before the end
     * TODO: Try template generated if tree that will use just one call with the correct number of left rows
     */
    int total_end = args.warp_right_end.y - (SHIFTS_PER_THREAD - 1);
    for (; warp_y_right < total_end; warp_y_right += 1) {
        compute_row_group<SHIFTS_PER_THREAD, 1, SHIFTS_PER_THREAD, 0>(
            warp,
            args,
            warp_y_right,
            args.warp_max_shift.y,
            sum
        );
    }

    wind_down<SHIFTS_PER_THREAD - 1, SHIFTS_PER_THREAD>(warp, args, sum);

    auto first_output_offset = args.output_pos.linear_idx(args.search_size.x);
    RES* matrix = args.out;

    // TODO: Maybe just check the x axis, Y axis should be filtered out by 0 NUM_RIGHT_ROWS
    if (args.output_pos.x < args.search_size.x && args.output_pos.y < args.search_size.y) {
        #pragma unroll
        for (dsize_t s = 0; s < SHIFTS_PER_THREAD; ++s) {
            auto output_offset = first_output_offset + s * args.search_size.x;
            if constexpr(ATOMIC) {
                atomicAdd(matrix + output_offset, sum[s]);
            } else {
                matrix[output_offset] = sum[s];
            }
        }
    }
}

template<dsize_t SHIFTS_PER_THREAD, dsize_t MAX_LEFT_ROWS, bool ATOMIC, dsize_t WARP_SIZE, typename T, typename RES>
__device__ void shuffle_multirow_both_impl_dispatch(
    const cg::thread_block_tile<WARP_SIZE>& warp,
    dsize_t num_thread_shifts,
    const warp_shuffle_impl_args<T, RES>& args
) {
    if constexpr(SHIFTS_PER_THREAD == 0) {
        // Zero is valid, if the warp is completely outside the result matrix

        // Silence the unused parameter warning
        (void)warp;
        (void)num_thread_shifts;
        (void)args;
    } else {
        if (SHIFTS_PER_THREAD == num_thread_shifts) {
            shuffle_multirow_both_impl<SHIFTS_PER_THREAD, MAX_LEFT_ROWS, ATOMIC>(
                warp,
                args
            );
        } else {
            shuffle_multirow_both_impl_dispatch<SHIFTS_PER_THREAD - 1, MAX_LEFT_ROWS, ATOMIC>(
                warp,
                num_thread_shifts,
                args
            );
        }
    }
}


/**
 * This kernel first computes the range which should be
 * computed by the current warp in the left and right matrices
 * and then always loads 32 values
 */
template<dsize_t MAX_SHIFTS_PER_THREAD, dsize_t MAX_LEFT_ROWS, typename T, typename RES>
__global__ void ccn_shuffle_multirow_both(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    // All warps of given block start at the same x, but each work on different row of output
    dsize2_t thread0_out_pos{
        ctb.group_index().x * ctb.group_dim().x,
        (ctb.group_index().y * ctb.group_dim().y + ctb.thread_index().y) * MAX_SHIFTS_PER_THREAD
    };
    dsize2_t last_warp_thread_out_pos = thread0_out_pos +
                                        dsize2_t{warp.size() - 1, 0};

    // Position in the output matrix
    // This is unique for each thread, as each thread computes a single shift which
    // corresponds to a single output value
    dsize2_t output_pos = thread0_out_pos +
                          dsize2_t{warp.thread_rank(), 0};

    dsize2_t half_search_size = (search_size - 1) / 2;

    // Min of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 0
    vec2<int> warp_min_shift = {
        static_cast<int>(thread0_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(thread0_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    // Max of the shifts computed by the threads of the current warp
    // This will always be the shift computed by thread 31 for the x axis
    //
    // It is clamped into search size as matrix may not be of size divisible by warp_size
    vec2<int> warp_max_shift = {
        static_cast<int>(min(last_warp_thread_out_pos.x, search_size.x - 1)) -
        static_cast<int>(half_search_size.x),
        // max_right_rows - 1 because + max_right_rows is the min_shift of next warp
        static_cast<int>(min(last_warp_thread_out_pos.y + MAX_SHIFTS_PER_THREAD - 1, search_size.y - 1)) -
        static_cast<int>(half_search_size.y)
    };


    // The start depends on the how far right the right matrix is shifted over the left matrix
    // if the right most shift, aka max shift is positive, that means that the left side of the right
    // matrix is inside the left matrix, so we need to start from the 0 element
    // if the max shift is negative, then absolute value tells us how many items of the right matrix are not needed
    // as they do not overlap in any shift computed by the matrix, as all smaller shifts have the right matrix more to the left
    // so they overlap less values
    dsize_t warp_x_right_start = warp_max_shift.x >= 0 ? 0 : -warp_max_shift.x;

    // The last value will be read by the min shift, so if it is larger than 0, the right side of the right matrix overhangs
    // the left matrix and so we don't need to reed the last abs(min_shift) values. Otherwise the right side of the right
    // matrix is inside the left matrix and we need to read it till the end.
    dsize_t warp_x_right_end = warp_min_shift.x >= 0 ? matrix_size.x - warp_min_shift.x : matrix_size.x;

    dsize_t warp_y_right_start = max(-warp_max_shift.y, 0);
    dsize_t warp_y_right_end = min(matrix_size.y - warp_min_shift.y, matrix_size.y);

    // Max shift might be smaller than min shift if warp is completely outside the out matrix
    // +1 because max_shift is inclusive, it is the last shift computed by this warp
    // so to get the number of shifts with both sides inclusive, we need to add 1
    auto num_thread_shifts = static_cast<dsize_t>(max(warp_max_shift.y - warp_min_shift.y + 1, 0));

    auto args = create_warp_shuffle_impl_args(
        left,
        right,
        out,
        dsize2_t{warp_x_right_start, warp_y_right_start},
        dsize2_t{warp_x_right_end, warp_y_right_end},
        warp_min_shift,
        warp_max_shift,
        output_pos,
        matrix_size,
        search_size
    );

    shuffle_multirow_both_impl_dispatch<MAX_SHIFTS_PER_THREAD, MAX_LEFT_ROWS, false>(
        warp,
        num_thread_shifts,
        args
    );
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_shuffle_multirow_both_kernel_args : public kernel_args {
public:
    dsize_t max_shifts_per_thread_;
    dsize_t max_left_rows_;

    ccn_shuffle_multirow_both_kernel_args(const ccn_shuffle_multirow_both_kernel_args&) = delete;
    ccn_shuffle_multirow_both_kernel_args& operator=(ccn_shuffle_multirow_both_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t max_shifts_per_thread,
        dsize_t max_left_rows
    ) {
        static ccn_shuffle_multirow_both_kernel_args instance;
        instance.set_common(block_size, grid_size, 0);
        instance.max_shifts_per_thread_ = max_shifts_per_thread;
        instance.max_left_rows_ = max_left_rows;
        set_last_kernel_launch_args(&instance);
    }

    [[nodiscard]] std::unordered_map<std::string, std::string> get_additional_args() const override {
        return std::unordered_map<std::string, std::string>{
            {"max_shifts_per_thread", std::to_string(max_shifts_per_thread_)},
            {"max_left_rows", std::to_string(max_left_rows_)}
        };
    }

private:
    ccn_shuffle_multirow_both_kernel_args()
        : kernel_args(),
          max_shifts_per_thread_(0),
          max_left_rows_(0)
    { }
};

template<dsize_t MAX_SHIFTS_PER_THREAD, dsize_t MAX_LEFT_ROWS, typename T, typename RES>
__host__ void ccn_shuffle_multirow_both_left_rows_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t warps_per_thread_block,
    dsize_t max_left_rows
) {
    if constexpr(MAX_LEFT_ROWS > 0) {
        if (MAX_LEFT_ROWS == max_left_rows) {
            dim3 num_threads(warp_size, warps_per_thread_block);
            dim3 num_blocks(
                div_up(search_size.x, num_threads.x),
                div_up(search_size.y, num_threads.y * MAX_SHIFTS_PER_THREAD)
            );

            ccn_shuffle_multirow_both<MAX_SHIFTS_PER_THREAD, MAX_LEFT_ROWS><<<num_blocks, num_threads>>>(
                left,
                right,
                out,
                matrix_size,
                search_size
            );

            ccn_shuffle_multirow_both_kernel_args::record_launch(
                num_threads,
                num_blocks,
                MAX_SHIFTS_PER_THREAD,
                MAX_LEFT_ROWS
            );
        } else {
            ccn_shuffle_multirow_both_left_rows_dispatch<MAX_SHIFTS_PER_THREAD, MAX_LEFT_ROWS - 1>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                warps_per_thread_block,
                max_left_rows
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)warps_per_thread_block;
        (void)max_left_rows;
        assert(false);
    }
}

template<dsize_t MAX_SHIFTS_PER_THREAD, dsize_t MAX_LEFT_ROWS, typename T, typename RES>
__host__ void ccn_shuffle_multirow_both_shifts_dispatch(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t warps_per_thread_block,
    dsize_t max_shifts_per_thread,
    dsize_t max_left_rows
) {
    if constexpr(MAX_SHIFTS_PER_THREAD > 0) {
        if (MAX_SHIFTS_PER_THREAD == max_shifts_per_thread) {
            ccn_shuffle_multirow_both_left_rows_dispatch<MAX_SHIFTS_PER_THREAD, MAX_LEFT_ROWS>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                warps_per_thread_block,
                max_left_rows
            );
        } else {
            ccn_shuffle_multirow_both_shifts_dispatch<MAX_SHIFTS_PER_THREAD - 1, MAX_LEFT_ROWS>(
                left,
                right,
                out,
                matrix_size,
                search_size,
                warps_per_thread_block,
                max_shifts_per_thread,
                max_left_rows
            );
        }
    } else {
        // TODO: Solve the -Wunused-but-set-parameter warning
        // Silence the confusing -Wunused-but-set-parameter warning
        // as we are not setting the parameters anywhere
        (void)left;
        (void)right;
        (void)out;
        (void)matrix_size;
        (void)search_size;
        (void)warps_per_thread_block;
        (void)max_shifts_per_thread;
        (void)max_left_rows;
        assert(false);
    }
}

} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_shuffle_multirow_both(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t warps_per_thread_block,
    dsize_t max_shifts_per_thread,
    dsize_t max_left_rows
) {
    if (warps_per_thread_block > 32) {
        throw std::runtime_error("Too many warps per thread block: "s + std::to_string(warps_per_thread_block) + " (max 32)");
    }

    if (max_shifts_per_thread > shifts_per_thread_limit) {
        throw std::runtime_error(
            "Too many shifts per thread: "s +
            std::to_string(max_shifts_per_thread) +
            "(max "s + std::to_string(shifts_per_thread_limit) +
            ")"s
        );
    }

    if (max_left_rows > left_rows_limit) {
        throw std::runtime_error(
            "Too many left rows per iteration: "s +
            std::to_string(max_left_rows) +
            "(max "s + std::to_string(left_rows_limit) +
            ")"s
        );
    }

    ccn_shuffle_multirow_both_shifts_dispatch<shifts_per_thread_limit, left_rows_limit>(
        left,
        right,
        out,
        matrix_size,
        search_size,
        warps_per_thread_block,
        max_shifts_per_thread,
        max_left_rows
    );
}

// template void run_ccn_shuffle_multirow_both<int, int>(
//         const int* __restrict__ left,
//         const int* __restrict__ right,
//         int* __restrict__ out,
//         dsize2_t matrix_size,
//         dsize2_t search_size,
//         dsize_t warps_per_thread_block,
//         dsize_t max_shifts_per_thread,
//         dsize_t max_left_rows
// );

template void run_ccn_shuffle_multirow_both<float, float>(
        const float* __restrict__ left,
        const float* __restrict__ right,
        float* __restrict__ out,
        dsize2_t matrix_size,
        dsize2_t search_size,
        dsize_t warps_per_thread_block,
        dsize_t max_shifts_per_thread,
        dsize_t max_left_rows
);

// template void run_ccn_shuffle_multirow_both<double, double>(
//         const double* __restrict__ left,
//         const double* __restrict__ right,
//         double* __restrict__ out,
//         dsize2_t matrix_size,
//         dsize2_t search_size,
//         dsize_t warps_per_thread_block,
//         dsize_t max_shifts_per_thread,
//         dsize_t max_left_rows
// );

}
