#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <stdexcept>

#include "types.cuh"
#include "shared_mem.cuh"
#include "warp_size.hpp"
#include "kernel_args.hpp"

namespace cg = cooperative_groups;

namespace cross {

namespace {

template<typename T, typename RES>
__global__ void ccn_block_per_shift(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size
) {
    cg::thread_block ctb = cg::this_thread_block();
    cg::thread_block_tile<warp_size> warp = cg::tiled_partition<warp_size>(ctb);

    dsize2_t block_out_pos{
        ctb.group_index().x,
        ctb.group_index().y
    };

    dsize2_t half_search_size = (search_size - 1) / 2;

    vec2<int> block_shift = {
        static_cast<int>(block_out_pos.x) - static_cast<int>(half_search_size.x),
        static_cast<int>(block_out_pos.y) - static_cast<int>(half_search_size.y)
    };

    dsize2_t right_start(
        max(0, -block_shift.x),
        max(0, -block_shift.y)
    );

    dsize2_t right_end(
        min(matrix_size.x - block_shift.x, matrix_size.x),
        min(matrix_size.y - block_shift.y, matrix_size.y)
    );

    dsize2_t overlap_size = right_end - right_start;
    dsize_t total_items = overlap_size.area();

    RES sum = 0;
    // Simpler internal loop, as is done in simple_indexing version,
    // leads to high thread divergence and much slower overall speed
    // so even though this is bottlenecked by the index computations,
    // it still runs much faster
    for (dsize_t i = ctb.thread_rank(); i < total_items; i += ctb.size()) {
        dsize_t overlap_row = i / overlap_size.x;
        dsize_t overlap_row_offset = i % overlap_size.x;

        dsize2_t right_idx = right_start + dsize2_t{overlap_row_offset, overlap_row};
        dsize2_t left_idx = dsize2_t{
            right_idx.x + block_shift.x,
            right_idx.y + block_shift.y
        };


        sum += left[left_idx.linear_idx(matrix_size.x)] * right[right_idx.linear_idx(matrix_size.x)];
    }

    // Reduce in each warp
    sum = cg::reduce(warp, sum, cg::plus<RES>());
    RES* shared = shared_memory_proxy<RES>();

    // First thread of each warp writes the warp sum into shared memory
    if (warp.thread_rank() == 0) {
        shared[warp.meta_group_rank()] = sum;
    }
    ctb.sync();

    // The first warp of the block reduces the values from shared memory into a single result
    if (warp.meta_group_rank() == 0) {
        // TODO: This expects max size of thread block to be 1024, which is true for all
        //  current compute capabilities
        if (warp.thread_rank() < warp.meta_group_size()) {
            sum = shared[warp.thread_rank()];
        } else {
            sum = 0;
        }
        sum = cg::reduce(warp, sum, cg::plus<RES>());

        if (warp.thread_rank() == 0) {
            out[block_out_pos.linear_idx(search_size.x)] = sum;
        }
    }
}

/**
 * Args used for the kernel call. The class is a singleton to minimize the impact
 * on measured time (prevent allocation etc.)
 */
class ccn_block_per_shift_kernel_args : public kernel_args {
public:
    ccn_block_per_shift_kernel_args(const ccn_block_per_shift_kernel_args&) = delete;
    ccn_block_per_shift_kernel_args& operator=(ccn_block_per_shift_kernel_args&) = delete;

    static void record_launch(
        dim3 block_size,
        dim3 grid_size,
        dsize_t shared_mem_bytes
    ) {
        static ccn_block_per_shift_kernel_args instance;
        instance.set_common(block_size, grid_size, shared_mem_bytes);
        set_last_kernel_launch_args(&instance);
    }
private:
    ccn_block_per_shift_kernel_args()
        : kernel_args()
    { }
};


} // END anonymous namespace

template<typename T, typename RES>
void run_ccn_block_per_shift(
    const T* __restrict__ left,
    const T* __restrict__ right,
    RES* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_block_size
) {
    if (cuda_block_size > 1024) {
        throw std::runtime_error("CUDA block too large: "s + std::to_string(cuda_block_size) + " (max 1024)");
    }

    dim3 num_threads(cuda_block_size);
    dim3 num_blocks(
        search_size.x,
        search_size.y
    );

    // One item for each wapr in a block
    dsize_t shared_mem_size = (cuda_block_size / warp_size) * sizeof(RES);
    ccn_block_per_shift<<<num_blocks, num_threads, shared_mem_size>>>(
        left,
        right,
        out,
        matrix_size,
        search_size
    );

    ccn_block_per_shift_kernel_args::record_launch(
        num_threads,
        num_blocks,
        shared_mem_size
    );
}

// template void run_ccn_block_per_shift<int, int>(
//     const int* __restrict__ left,
//     const int* __restrict__ right,
//     int* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t cuda_block_size
// );

template void run_ccn_block_per_shift<float, float>(
    const float* __restrict__ left,
    const float* __restrict__ right,
    float* __restrict__ out,
    dsize2_t matrix_size,
    dsize2_t search_size,
    dsize_t cuda_block_size
);

// template void run_ccn_block_per_shift<double, double>(
//     const double* __restrict__ left,
//     const double* __restrict__ right,
//     double* __restrict__ out,
//     dsize2_t matrix_size,
//     dsize2_t search_size,
//     dsize_t cuda_block_size
// );

}
